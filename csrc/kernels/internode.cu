#include "hip/hip_runtime.h"
#include "configs.cuh"
#include "buffer.cuh"
#include "exception.cuh"
#include "launch.cuh"
#include "utils.cuh"
#include "ibgda_device.cuh"

namespace deep_ep {

namespace internode {

extern nvshmem_team_t cpu_rdma_team;

struct SourceMeta {
    int src_rdma_rank, is_token_in_nvl_rank_bits;

    EP_STATIC_ASSERT(NUM_MAX_NVL_PEERS == 8, "Invalid number of maximum NVL peers");

    __forceinline__ SourceMeta() = default;

    // TODO: faster encoding
    __device__ __forceinline__ SourceMeta(int rdma_rank, const bool* is_token_in_nvl_ranks) {
        src_rdma_rank = rdma_rank;
        is_token_in_nvl_rank_bits = is_token_in_nvl_ranks[0];
        #pragma unroll
        for (int i = 1; i < NUM_MAX_NVL_PEERS; ++ i)
            is_token_in_nvl_rank_bits |= is_token_in_nvl_ranks[i] << i;
    }

    __device__ __forceinline__ bool is_token_in_nvl_rank(int nvl_rank) const {
        return (is_token_in_nvl_rank_bits >> nvl_rank) & 1;
    }
};

EP_STATIC_ASSERT(sizeof(SourceMeta) % sizeof(int) == 0, "Invalid size of `SourceMeta`");

int get_source_meta_bytes() {
    return sizeof(SourceMeta);
}

__host__ __device__ __forceinline__
int get_num_bytes_per_rdma_token(int hidden_int4, int num_scales, int num_topk_idx, int num_topk_weights) {
    return static_cast<int>(align(hidden_int4 * sizeof(int4) + sizeof(SourceMeta) + num_scales * sizeof(float) + num_topk_idx * sizeof(int) + num_topk_weights * sizeof(float), sizeof(int4)));
}

__host__ __device__ __forceinline__
std::pair<int, int> get_rdma_clean_meta(int hidden_int4, int num_scales, int num_topk_idx, int num_topk_weights, int num_rdma_ranks, int num_rdma_recv_buffer_tokens, int num_sms) {
    // Return `int32_t` offset and count to clean
    return {
        (get_num_bytes_per_rdma_token(hidden_int4, num_scales, num_topk_idx, num_topk_weights) * num_rdma_recv_buffer_tokens * num_rdma_ranks * 2 * num_sms) / sizeof(int),
        (NUM_MAX_NVL_PEERS * 2 + 4) * num_rdma_ranks * 2 * num_sms
    };
}

__host__ __device__ __forceinline__
std::pair<int, int> get_nvl_clean_meta(int hidden_int4, int num_scales, int num_topk_idx, int num_topk_weights, int num_rdma_ranks, int num_nvl_ranks, int num_nvl_recv_buffer_tokens, int num_sms) {
    // Return `int32_t` offset and to clean
    EP_STATIC_ASSERT(sizeof(SourceMeta) % sizeof(int) == 0, "Invalid size of `SourceMeta`");
    return {
        (num_nvl_recv_buffer_tokens * (hidden_int4 * sizeof(int4) + num_scales * sizeof(float) + num_topk_idx * sizeof(int) + num_topk_weights * sizeof(float) + sizeof(SourceMeta)) * num_nvl_ranks * num_sms) / sizeof(int),
        num_nvl_ranks * (2 * num_rdma_ranks + 2) * num_sms,
    };
}

template <bool kLowLatencyMode>
__forceinline__ __device__ int translate_dst_rdma_rank(const int dst_rdma_rank, const int nvl_rank) {
    return kLowLatencyMode ? (dst_rdma_rank * NUM_MAX_NVL_PEERS + nvl_rank) : dst_rdma_rank;
}

template <bool kLowLatencyMode>
__forceinline__ __device__ void nvshmem_sync_with_same_gpu_idx(const nvshmem_team_t& rdma_team) {
    kLowLatencyMode ? void(nvshmem_sync(rdma_team)) : nvshmem_sync_all();
}

template <bool kLowLatencyMode, int kNumRDMARanks>
__global__ void
notify_dispatch(const int* num_tokens_per_rank, int* moe_recv_counter_mapped, int num_ranks,
                const int* num_tokens_per_rdma_rank, int* moe_recv_rdma_counter_mapped,
                const int* num_tokens_per_expert, int* moe_recv_expert_counter_mapped, int num_experts,
                const bool* is_token_in_rank, int num_tokens, int num_channels, int expert_alignment,
                const int rdma_clean_offset, const int rdma_num_int_clean,
                const int nvl_clean_offset, const int nvl_num_int_clean,
                int* rdma_channel_prefix_matrix, int* recv_rdma_rank_prefix_sum,
                int* gbl_channel_prefix_matrix, int* recv_gbl_rank_prefix_sum,
                void* rdma_buffer_ptr,
                void** buffer_ptrs, int** barrier_signal_ptrs, int rank,
                const nvshmem_team_t rdma_team) {
    auto sm_id = static_cast<int>(blockIdx.x);
    auto thread_id = static_cast<int>(threadIdx.x), warp_id = thread_id / 32, lane_id = get_lane_id();
    auto num_threads = static_cast<int>(blockDim.x), num_warps = num_threads / 32;

    auto rdma_rank = rank / NUM_MAX_NVL_PEERS, nvl_rank = rank % NUM_MAX_NVL_PEERS;
    auto num_rdma_experts = num_experts / kNumRDMARanks, num_nvl_experts = num_rdma_experts / NUM_MAX_NVL_PEERS;

    if (sm_id == 0) {
        // Communication with others
        // Global barrier: the first warp does intra-node sync, the second warp does internode sync
        EP_DEVICE_ASSERT(num_warps > 1);
        EP_DEVICE_ASSERT(kNumRDMARanks <= num_threads);
        if (thread_id == 32)
            nvshmem_sync_with_same_gpu_idx<kLowLatencyMode>(rdma_team);
        barrier_block<NUM_MAX_NVL_PEERS, true>(barrier_signal_ptrs, nvl_rank);

        // Send numbers of tokens per rank/expert to RDMA ranks
        auto rdma_buffer_ptr_int = static_cast<int*>(rdma_buffer_ptr);
        auto rdma_recv_num_tokens_mixed = SymBuffer<int>(rdma_buffer_ptr, NUM_MAX_NVL_PEERS + num_rdma_experts + 1, kNumRDMARanks);

        // Clean up for later data dispatch
        EP_DEVICE_ASSERT(rdma_recv_num_tokens_mixed.total_bytes <= rdma_clean_offset * sizeof(int));
        #pragma unroll
        for (int i = thread_id; i < rdma_num_int_clean; i += num_threads)
            rdma_buffer_ptr_int[rdma_clean_offset + i] = 0;

        // Copy to send buffer
        #pragma unroll
        for (int i = thread_id; i < num_ranks; i += num_threads)
            rdma_recv_num_tokens_mixed.send_buffer(i / NUM_MAX_NVL_PEERS)[i % NUM_MAX_NVL_PEERS] = num_tokens_per_rank[i];
        #pragma unroll
        for (int i = thread_id; i < num_experts; i += num_threads)
            rdma_recv_num_tokens_mixed.send_buffer(i / num_rdma_experts)[NUM_MAX_NVL_PEERS + i % num_rdma_experts] = num_tokens_per_expert[i];
        if (thread_id < kNumRDMARanks)
            rdma_recv_num_tokens_mixed.send_buffer(thread_id)[NUM_MAX_NVL_PEERS + num_rdma_experts] = num_tokens_per_rdma_rank[thread_id];
        __syncthreads();

        // Issue send
        // TODO: more light fence or barrier or signaling
        // TODO: overlap EP barrier and NVL cleaning
        for (int i = warp_id; i < kNumRDMARanks; i += num_warps) {
            if (i != rdma_rank) {
                nvshmemi_ibgda_put_nbi_warp<true>(reinterpret_cast<uint64_t>(rdma_recv_num_tokens_mixed.recv_buffer(rdma_rank)),
                                                reinterpret_cast<uint64_t>(rdma_recv_num_tokens_mixed.send_buffer(i)),
                                                (NUM_MAX_NVL_PEERS + num_rdma_experts + 1) * sizeof(int),
                                                translate_dst_rdma_rank<kLowLatencyMode>(i, nvl_rank), 0, lane_id, 0);
            } else { 
                UNROLLED_WARP_COPY(1, lane_id, NUM_MAX_NVL_PEERS + num_rdma_experts + 1, 
                                    rdma_recv_num_tokens_mixed.recv_buffer(rdma_rank), 
                                    rdma_recv_num_tokens_mixed.send_buffer(i), 
                                    ld_volatile_global, st_na_global);
            }
        }
        
        __syncthreads();
        if (thread_id < kNumRDMARanks and thread_id != rdma_rank)
            nvshmemi_ibgda_quiet(translate_dst_rdma_rank<kLowLatencyMode>(thread_id, nvl_rank), 0);

        __syncthreads();
        if (thread_id == 0)
            nvshmem_sync_with_same_gpu_idx<kLowLatencyMode>(rdma_team);
        __syncthreads();

        // NVL buffers
        auto nvl_send_buffer = thread_id < NUM_MAX_NVL_PEERS ? buffer_ptrs[thread_id] : nullptr;
        auto nvl_recv_buffer = buffer_ptrs[nvl_rank];
        auto nvl_reduced_num_tokens_per_expert = Buffer<int>(nvl_recv_buffer, num_rdma_experts).advance_also(nvl_send_buffer);
        auto nvl_send_num_tokens_per_rank = AsymBuffer<int>(nvl_send_buffer, kNumRDMARanks, NUM_MAX_NVL_PEERS);
        auto nvl_send_num_tokens_per_expert = AsymBuffer<int>(nvl_send_buffer, num_nvl_experts, NUM_MAX_NVL_PEERS);
        auto nvl_recv_num_tokens_per_rank = AsymBuffer<int>(nvl_recv_buffer, kNumRDMARanks, NUM_MAX_NVL_PEERS);
        auto nvl_recv_num_tokens_per_expert = AsymBuffer<int>(nvl_recv_buffer, num_nvl_experts, NUM_MAX_NVL_PEERS);

        // Clean up for later data dispatch
        auto nvl_buffer_ptr_int = static_cast<int*>(buffer_ptrs[nvl_rank]);
        EP_DEVICE_ASSERT(nvl_reduced_num_tokens_per_expert.total_bytes + nvl_send_num_tokens_per_rank.total_bytes +
                         nvl_send_num_tokens_per_expert.total_bytes <= nvl_clean_offset * sizeof(int));
        #pragma unroll
        for (int i = thread_id; i < nvl_num_int_clean; i += num_threads)
            nvl_buffer_ptr_int[nvl_clean_offset + i] = 0;

        // Reduce number of tokens per expert into the NVL send buffer
        // TODO: may use NVSHMEM reduction
        EP_DEVICE_ASSERT(num_rdma_experts <= num_threads);
        if (thread_id < num_rdma_experts) {
            int sum = 0;
            #pragma unroll
            for (int i = 0; i < kNumRDMARanks; ++ i)
                sum += rdma_recv_num_tokens_mixed.recv_buffer(i)[NUM_MAX_NVL_PEERS + thread_id];
            nvl_reduced_num_tokens_per_expert[thread_id] = sum;
        }
        __syncthreads();

        // Reduce RDMA received tokens
        if (thread_id == 0) {
            int sum = 0;
            #pragma unroll
            for (int i = 0; i < kNumRDMARanks; ++ i) {
                sum += rdma_recv_num_tokens_mixed.recv_buffer(i)[NUM_MAX_NVL_PEERS + num_rdma_experts];
                recv_rdma_rank_prefix_sum[i] = sum;
            }
            while (ld_volatile_global(moe_recv_rdma_counter_mapped) != -1);
            *moe_recv_rdma_counter_mapped = sum;
        }

        // Send numbers of tokens per rank/expert to NVL ranks
        EP_DEVICE_ASSERT(NUM_MAX_NVL_PEERS <= num_threads);
        if (thread_id < NUM_MAX_NVL_PEERS) {
            #pragma unroll
            for (int i = 0; i < kNumRDMARanks; ++ i)
                nvl_send_num_tokens_per_rank.buffer(nvl_rank)[i] = rdma_recv_num_tokens_mixed.recv_buffer(i)[thread_id];
            #pragma unroll
            for (int i = 0; i < num_nvl_experts; ++ i)
                nvl_send_num_tokens_per_expert.buffer(nvl_rank)[i] = nvl_reduced_num_tokens_per_expert[thread_id * num_nvl_experts + i];
        }
        barrier_block<NUM_MAX_NVL_PEERS>(barrier_signal_ptrs, nvl_rank);

        // Reduce the number of tokens per rank/expert
        EP_DEVICE_ASSERT(num_nvl_experts <= num_threads);
        if (thread_id == 0) {
            int sum = 0;
            #pragma unroll
            for (int i = 0; i < num_ranks; ++ i) {
                int src_rdma_rank = i / NUM_MAX_NVL_PEERS, src_nvl_rank = i % NUM_MAX_NVL_PEERS;
                sum += nvl_recv_num_tokens_per_rank.buffer(src_nvl_rank)[src_rdma_rank];
                recv_gbl_rank_prefix_sum[i] = sum;
            }
            while (ld_volatile_global(moe_recv_counter_mapped) != -1);
            *moe_recv_counter_mapped = sum;
        }
        if (thread_id < num_nvl_experts) {
            int sum = 0;
            #pragma unroll
            for (int i = 0; i < NUM_MAX_NVL_PEERS; ++ i)
                sum += nvl_recv_num_tokens_per_expert.buffer(i)[thread_id];
            sum = (sum + expert_alignment - 1) / expert_alignment * expert_alignment;
            while (ld_volatile_global(moe_recv_expert_counter_mapped + thread_id) != -1);
            moe_recv_expert_counter_mapped[thread_id] = sum;
        }

        // Finally barrier
        if (thread_id == 32)
            nvshmem_sync_with_same_gpu_idx<kLowLatencyMode>(rdma_team);
        barrier_block<NUM_MAX_NVL_PEERS>(barrier_signal_ptrs, nvl_rank);
    } else {
        // Calculate meta data
        int dst_rdma_rank = sm_id - 1;
        for (int channel_id = warp_id; channel_id < num_channels; channel_id += num_warps) {
            int token_start_idx, token_end_idx;
            get_channel_task_range(num_tokens, num_channels, channel_id, token_start_idx, token_end_idx);

            // Iterate over tokens
            int total_count = 0, per_nvl_rank_count[NUM_MAX_NVL_PEERS] = {0};
            for (int64_t i = token_start_idx + lane_id; i < token_end_idx; i += 32) {
                EP_STATIC_ASSERT(NUM_MAX_NVL_PEERS * sizeof(bool) == sizeof(uint64_t), "Invalid number of NVL peers");
                auto is_token_in_rank_uint64 = *reinterpret_cast<const uint64_t*>(is_token_in_rank + i * num_ranks + dst_rdma_rank * NUM_MAX_NVL_PEERS);
                auto is_token_in_rank_values = reinterpret_cast<const bool*>(&is_token_in_rank_uint64);
                #pragma unroll
                for (int j = 0; j < NUM_MAX_NVL_PEERS; ++ j)
                    per_nvl_rank_count[j] += is_token_in_rank_values[j];
                total_count += (is_token_in_rank_uint64 != 0);
            }

            // Warp reduce
            total_count = warp_reduce_sum(total_count);
            #pragma unroll
            for (int i = 0; i < NUM_MAX_NVL_PEERS; ++ i)
                per_nvl_rank_count[i] = warp_reduce_sum(per_nvl_rank_count[i]);

            // Write into channel matrix
            if (lane_id == 0) {
                #pragma unroll
                for (int i = 0; i < NUM_MAX_NVL_PEERS; ++ i)
                    gbl_channel_prefix_matrix[(dst_rdma_rank * NUM_MAX_NVL_PEERS + i) * num_channels + channel_id] = per_nvl_rank_count[i];
                rdma_channel_prefix_matrix[dst_rdma_rank * num_channels + channel_id] = total_count;
            }
        }

        // Calculate prefix sum
        __syncthreads();
        if (thread_id == 0) {
            auto prefix_row = rdma_channel_prefix_matrix + dst_rdma_rank * num_channels;
            #pragma unroll
            for (int i = 1; i < num_channels; ++ i)
                prefix_row[i] += prefix_row[i - 1];
        }

        EP_STATIC_ASSERT(NUM_MAX_NVL_PEERS <= 32, "Invalid number of NVL peers");
        if (thread_id < NUM_MAX_NVL_PEERS) {
            auto prefix_row = gbl_channel_prefix_matrix + (dst_rdma_rank * NUM_MAX_NVL_PEERS + thread_id) * num_channels;
            #pragma unroll
            for (int i = 1; i < num_channels; ++ i)
                prefix_row[i] += prefix_row[i - 1];
        }
    }
}

void notify_dispatch(const int* num_tokens_per_rank, int* moe_recv_counter_mapped, int num_ranks,
                     const int* num_tokens_per_rdma_rank, int* moe_recv_rdma_counter_mapped,
                     const int* num_tokens_per_expert, int* moe_recv_expert_counter_mapped, int num_experts,
                     const bool* is_token_in_rank, int num_tokens, int num_channels,
                     int hidden_int4, int num_scales, int num_topk, int expert_alignment,
                     int* rdma_channel_prefix_matrix, int* recv_rdma_rank_prefix_sum,
                     int* gbl_channel_prefix_matrix, int* recv_gbl_rank_prefix_sum,
                     void* rdma_buffer_ptr, int num_max_rdma_chunked_recv_tokens,
                     void** buffer_ptrs, int num_max_nvl_chunked_recv_tokens,
                     int** barrier_signal_ptrs, int rank,
                     hipStream_t stream, int64_t num_rdma_bytes, int64_t num_nvl_bytes,
                     bool low_latency_mode) {
#define NOTIFY_DISPATCH_LAUNCH_CASE(num_rdma_ranks) { \
    auto notify_dispatch_func = low_latency_mode ? \
        notify_dispatch<true, num_rdma_ranks> : notify_dispatch<false, num_rdma_ranks>; \
    LAUNCH_KERNEL(&cfg, notify_dispatch_func, \
                  num_tokens_per_rank, moe_recv_counter_mapped, num_ranks, \
                  num_tokens_per_rdma_rank, moe_recv_rdma_counter_mapped, \
                  num_tokens_per_expert, moe_recv_expert_counter_mapped, num_experts, \
                  is_token_in_rank, num_tokens, num_channels, expert_alignment, \
                  rdma_clean_meta.first, rdma_clean_meta.second, \
                  nvl_clean_meta.first, nvl_clean_meta.second, \
                  rdma_channel_prefix_matrix, recv_rdma_rank_prefix_sum, \
                  gbl_channel_prefix_matrix, recv_gbl_rank_prefix_sum, \
                  rdma_buffer_ptr, \
                  buffer_ptrs, barrier_signal_ptrs, rank, \
                  cpu_rdma_team); } break

    constexpr int kNumThreads = 512;
    const auto num_rdma_ranks = num_ranks / NUM_MAX_NVL_PEERS;

    // Get clean meta
    auto rdma_clean_meta = get_rdma_clean_meta(hidden_int4, num_scales, num_topk, num_topk, num_rdma_ranks, num_max_rdma_chunked_recv_tokens, num_channels);
    auto nvl_clean_meta = get_nvl_clean_meta(hidden_int4, num_scales, num_topk, num_topk, num_rdma_ranks, NUM_MAX_NVL_PEERS, num_max_nvl_chunked_recv_tokens, num_channels);
    EP_HOST_ASSERT((rdma_clean_meta.first + rdma_clean_meta.second) * sizeof(int) <= num_rdma_bytes);
    EP_HOST_ASSERT((nvl_clean_meta.first + nvl_clean_meta.second) * sizeof(int) <= num_nvl_bytes);
    EP_HOST_ASSERT(num_rdma_bytes < std::numeric_limits<int>::max());
    EP_HOST_ASSERT(num_nvl_bytes < std::numeric_limits<int>::max());

    // Launch kernel
    SETUP_LAUNCH_CONFIG(1 + num_rdma_ranks, kNumThreads, stream);
    SWITCH_RDMA_RANKS(NOTIFY_DISPATCH_LAUNCH_CASE);
#undef NOTIFY_DISPATCH_LAUNCH_CASE
}

// At most 8 RDMA ranks to be sent
constexpr int get_num_topk_rdma_ranks(int num_rdma_ranks) {
    return num_rdma_ranks < 8 ? num_rdma_ranks : 8;
}

template <bool kLowLatencyMode, int kNumRDMARanks, bool kCachedMode,
          int kNumDispatchRDMASenderWarps, int kNumTopkRDMARanks = get_num_topk_rdma_ranks(kNumRDMARanks)>
__global__ void __launch_bounds__(((kNumDispatchRDMASenderWarps + 1 + NUM_MAX_NVL_PEERS) * 32), 1)
dispatch(int4* recv_x, float* recv_x_scales, int* recv_topk_idx, float* recv_topk_weights, SourceMeta* recv_src_meta,
         const int4* x, const float* x_scales, const int* topk_idx, const float* topk_weights,
         int* send_rdma_head, int* send_nvl_head,
         int* recv_rdma_channel_prefix_matrix, int* recv_gbl_channel_prefix_matrix,
         const int* rdma_channel_prefix_matrix, const int* recv_rdma_rank_prefix_sum,
         const int* gbl_channel_prefix_matrix, const int* recv_gbl_rank_prefix_sum,
         const bool* is_token_in_rank,
         int num_tokens, int hidden_int4, int num_scales, int num_topk, int num_experts, int global_expert_id_offset,
         int scale_token_stride, int scale_hidden_stride,
         void* rdma_buffer_ptr, int num_max_rdma_chunked_send_tokens, int num_max_rdma_chunked_recv_tokens,
         void** buffer_ptrs, int num_max_nvl_chunked_send_tokens, int num_max_nvl_chunked_recv_tokens,
         int rank, int num_ranks) {
    enum class WarpRole {
        kRDMASender,
        kRDMASenderCoordinator,
        kRDMAAndNVLForwarder,
        kForwarderCoordinator,
        kNVLReceivers
    };

    const auto num_sms = static_cast<int>(gridDim.x);
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto num_threads = static_cast<int>(blockDim.x), num_warps = num_threads / 32;
    const auto thread_id = static_cast<int>(threadIdx.x), warp_id = thread_id / 32, lane_id = get_lane_id();
    const auto num_channels = num_sms / 2, channel_id = sm_id / 2;
    const bool is_forwarder = sm_id % 2 == 0;
    const auto rdma_rank = rank / NUM_MAX_NVL_PEERS, nvl_rank = rank % NUM_MAX_NVL_PEERS;

    EP_DEVICE_ASSERT(ibgda_get_state()->num_rc_per_pe == num_channels or ibgda_get_state()->num_rc_per_pe >= num_sms);

    const auto role_meta = [=]() -> std::pair<WarpRole, int> {
        if (is_forwarder) {
            if (warp_id < NUM_MAX_NVL_PEERS) {
                return {WarpRole::kRDMAAndNVLForwarder, (warp_id + channel_id) % NUM_MAX_NVL_PEERS};
            } else {
                return {WarpRole::kForwarderCoordinator, warp_id - NUM_MAX_NVL_PEERS};
            }
        } else if (warp_id < kNumDispatchRDMASenderWarps) {
            return {WarpRole::kRDMASender, -1};
        } else if (warp_id == kNumDispatchRDMASenderWarps) {
            return {WarpRole::kRDMASenderCoordinator, -1};
        } else {
            return {WarpRole::kNVLReceivers, (warp_id + channel_id - kNumDispatchRDMASenderWarps) % NUM_MAX_NVL_PEERS};
        }
    }();
    auto warp_role = role_meta.first;
    auto target_rank = role_meta.second; // Not applicable for RDMA senders
    EP_DEVICE_ASSERT(num_warps == kNumDispatchRDMASenderWarps + 1 + NUM_MAX_NVL_PEERS);

    // Data checks
    EP_DEVICE_ASSERT(num_topk <= 32);

    // RDMA symmetric layout
    EP_STATIC_ASSERT(NUM_MAX_NVL_PEERS * sizeof(bool) == sizeof(uint64_t), "Invalid number of NVL peers");
    auto hidden_bytes = hidden_int4 * sizeof(int4);
    auto num_bytes_per_rdma_token = get_num_bytes_per_rdma_token(hidden_int4, num_scales, num_topk, num_topk);
    auto rdma_channel_data = SymBuffer<int8_t>(rdma_buffer_ptr, num_max_rdma_chunked_recv_tokens * num_bytes_per_rdma_token, kNumRDMARanks, channel_id, num_channels);
    auto rdma_channel_meta = SymBuffer<int>(rdma_buffer_ptr, NUM_MAX_NVL_PEERS * 2 + 2, kNumRDMARanks, channel_id, num_channels);
    auto rdma_channel_head = SymBuffer<uint64_t, false>(rdma_buffer_ptr, 1, kNumRDMARanks, channel_id, num_channels);
    auto rdma_channel_tail = SymBuffer<uint64_t, false>(rdma_buffer_ptr, 1, kNumRDMARanks, channel_id, num_channels);

    // NVL buffer layouts
    // NOTES: `rs_wr_buffer_ptr` means "Read for Senders, Write for Receivers", `ws_rr_buffer_ptr` means "Write for Senders, Read for Receivers"
    void *rs_wr_buffer_ptr = nullptr, *ws_rr_buffer_ptr = nullptr;
    int rs_wr_rank = 0, ws_rr_rank = 0;
    if (warp_role == WarpRole::kRDMAAndNVLForwarder)
        rs_wr_buffer_ptr = buffer_ptrs[nvl_rank], ws_rr_buffer_ptr = buffer_ptrs[target_rank], rs_wr_rank = nvl_rank, ws_rr_rank = target_rank;
    if (warp_role == WarpRole::kNVLReceivers)
        rs_wr_buffer_ptr = buffer_ptrs[target_rank], ws_rr_buffer_ptr = buffer_ptrs[nvl_rank], rs_wr_rank = target_rank, ws_rr_rank = nvl_rank;

    // Allocate buffers
    auto nvl_channel_x = AsymBuffer<int4>(ws_rr_buffer_ptr, num_max_nvl_chunked_recv_tokens * hidden_int4, NUM_MAX_NVL_PEERS, channel_id, num_channels, rs_wr_rank).advance_also(rs_wr_buffer_ptr);
    auto nvl_channel_src_meta = AsymBuffer<SourceMeta>(ws_rr_buffer_ptr, num_max_nvl_chunked_recv_tokens, NUM_MAX_NVL_PEERS, channel_id, num_channels, rs_wr_rank).advance_also(rs_wr_buffer_ptr);
    auto nvl_channel_x_scales = AsymBuffer<float>(ws_rr_buffer_ptr, num_max_nvl_chunked_recv_tokens * num_scales, NUM_MAX_NVL_PEERS, channel_id, num_channels, rs_wr_rank).advance_also(rs_wr_buffer_ptr);
    auto nvl_channel_topk_idx = AsymBuffer<int>(ws_rr_buffer_ptr, num_max_nvl_chunked_recv_tokens * num_topk, NUM_MAX_NVL_PEERS, channel_id, num_channels, rs_wr_rank).advance_also(rs_wr_buffer_ptr);
    auto nvl_channel_topk_weights = AsymBuffer<float>(ws_rr_buffer_ptr, num_max_nvl_chunked_recv_tokens * num_topk, NUM_MAX_NVL_PEERS, channel_id, num_channels, rs_wr_rank).advance_also(rs_wr_buffer_ptr);
    auto nvl_channel_prefix_start = AsymBuffer<int>(ws_rr_buffer_ptr, kNumRDMARanks, NUM_MAX_NVL_PEERS, channel_id, num_channels, rs_wr_rank).advance_also(rs_wr_buffer_ptr);
    auto nvl_channel_prefix_end = AsymBuffer<int>(ws_rr_buffer_ptr, kNumRDMARanks, NUM_MAX_NVL_PEERS, channel_id, num_channels, rs_wr_rank).advance_also(rs_wr_buffer_ptr);
    auto nvl_channel_head = AsymBuffer<int>(rs_wr_buffer_ptr, 1, NUM_MAX_NVL_PEERS, channel_id, num_channels, ws_rr_rank).advance_also(ws_rr_buffer_ptr);
    auto nvl_channel_tail = AsymBuffer<int>(ws_rr_buffer_ptr, 1, NUM_MAX_NVL_PEERS, channel_id, num_channels, rs_wr_rank).advance_also(rs_wr_buffer_ptr);

    // RDMA sender warp synchronization
    // NOTES: `rdma_send_channel_tail` means the latest released tail
    // NOTES: `rdma_send_channel_window` means the ongoing 32 transactions' status
    __shared__ int rdma_send_channel_lock[kNumRDMARanks];
    __shared__ int rdma_send_channel_tail[kNumRDMARanks];
    __shared__ uint32_t rdma_send_channel_window[kNumRDMARanks];
    auto sync_rdma_sender_smem = []() { asm volatile("bar.sync 0, %0;" :: "r"((kNumDispatchRDMASenderWarps + 1) * 32)); };

    // Forward warp synchronization
    __shared__ volatile int forward_channel_head[NUM_MAX_NVL_PEERS][kNumRDMARanks];
    __shared__ volatile bool forward_channel_retired[NUM_MAX_NVL_PEERS];
    auto sync_forwarder_smem = []() { asm volatile("bar.sync 1, %0;" :: "r"((NUM_MAX_NVL_PEERS + 1) * 32)); };

    if (warp_role == WarpRole::kRDMASender) {
        // Get tasks
        int token_start_idx, token_end_idx;
        get_channel_task_range(num_tokens, num_channels, channel_id, token_start_idx, token_end_idx);

        // Send number of tokens in this channel by `-value - 1`
        EP_STATIC_ASSERT(NUM_MAX_NVL_PEERS * 2 + 2 <= 32, "Invalid number of NVL peers");
        for (int dst_rdma_rank = warp_id; dst_rdma_rank < kNumRDMARanks; dst_rdma_rank += kNumDispatchRDMASenderWarps) {
            auto dst_ptr = dst_rdma_rank == rdma_rank ? rdma_channel_meta.recv_buffer(dst_rdma_rank) : rdma_channel_meta.send_buffer(dst_rdma_rank);
            if (lane_id < NUM_MAX_NVL_PEERS) {
                dst_ptr[lane_id] = -(channel_id == 0 ? 0 : gbl_channel_prefix_matrix[(dst_rdma_rank * NUM_MAX_NVL_PEERS + lane_id) * num_channels + channel_id - 1]) - 1;
            } else if (lane_id < NUM_MAX_NVL_PEERS * 2) {
                dst_ptr[lane_id] = -gbl_channel_prefix_matrix[(dst_rdma_rank * NUM_MAX_NVL_PEERS + lane_id - NUM_MAX_NVL_PEERS) * num_channels + channel_id] - 1;
            } else if (lane_id == NUM_MAX_NVL_PEERS * 2) {
                dst_ptr[lane_id] = -(channel_id == 0 ? 0 : rdma_channel_prefix_matrix[dst_rdma_rank * num_channels + channel_id - 1]) - 1;
            } else if (lane_id == NUM_MAX_NVL_PEERS * 2 + 1) {
                dst_ptr[lane_id] = -rdma_channel_prefix_matrix[dst_rdma_rank * num_channels + channel_id] - 1;
            }
            __syncwarp();

            // Issue RDMA for non-local ranks
            if (dst_rdma_rank != rdma_rank) {
                nvshmemi_ibgda_put_nbi_warp<true>(reinterpret_cast<uint64_t>(rdma_channel_meta.recv_buffer(rdma_rank)),
                                                  reinterpret_cast<uint64_t>(rdma_channel_meta.send_buffer(dst_rdma_rank)),
                                                  sizeof(int) * (NUM_MAX_NVL_PEERS * 2 + 2),
                                                  translate_dst_rdma_rank<kLowLatencyMode>(dst_rdma_rank, nvl_rank),
                                                  channel_id, lane_id, 0);
            }
        }
        sync_rdma_sender_smem();

        // Iterate over tokens and copy into buffer
        int64_t token_idx;
        int cached_rdma_channel_head = 0, global_rdma_tail_idx = 0;
        auto send_buffer = lane_id == rdma_rank ? rdma_channel_data.recv_buffer(lane_id) : rdma_channel_data.send_buffer(lane_id);
        for (token_idx = token_start_idx; token_idx < token_end_idx; ++ token_idx) {
            // Read RDMA rank existence
            uint64_t is_token_in_rank_uint64 = 0;
            if (lane_id < kNumRDMARanks) {
                is_token_in_rank_uint64 = __ldg(reinterpret_cast<const uint64_t*>(is_token_in_rank + token_idx * num_ranks + lane_id * NUM_MAX_NVL_PEERS));
                global_rdma_tail_idx += (is_token_in_rank_uint64 != 0);
            }
            __syncwarp();

            // Skip the token which does not belong to this warp
            if ((token_idx - token_start_idx) % kNumDispatchRDMASenderWarps != warp_id)
                continue;
            auto rdma_tail_idx = is_token_in_rank_uint64 == 0 ? -1 : global_rdma_tail_idx - 1;

            // Wait the remote buffer to be released
            auto start_time = clock64();
            while (is_token_in_rank_uint64 != 0 and rdma_tail_idx - cached_rdma_channel_head >= num_max_rdma_chunked_recv_tokens) {
                cached_rdma_channel_head = static_cast<int>(ld_volatile_global(rdma_channel_head.buffer(lane_id)));

                // Timeout check
                if (clock64() - start_time >= NUM_TIMEOUT_CYCLES) {
                    printf("DeepEP dispatch RDMA sender timeout, channel: %d, RDMA: %d, nvl: %d, dst RDMA lane: %d, head: %d, tail: %d\n",
                           channel_id, rdma_rank, nvl_rank, lane_id, cached_rdma_channel_head, rdma_tail_idx);
                    trap();
                }
            }
            __syncwarp();

            // Store RDMA head for combine
            if (lane_id < kNumRDMARanks and not kCachedMode)
                send_rdma_head[token_idx * kNumRDMARanks + lane_id] = rdma_tail_idx;

            // Broadcast tails
            SourceMeta src_meta;
            int num_topk_ranks = 0, topk_ranks[kNumTopkRDMARanks];
            void* dst_send_buffers[kNumTopkRDMARanks];
            #pragma unroll
            for (int i = 0, slot_idx; i < kNumRDMARanks; ++ i) if ((slot_idx = __shfl_sync(0xffffffff, rdma_tail_idx, i)) >= 0) {
                slot_idx = slot_idx % num_max_rdma_chunked_recv_tokens;
                topk_ranks[num_topk_ranks] = i;
                auto recv_is_token_in_rank_uint64 = broadcast(is_token_in_rank_uint64, i);
                auto recv_is_token_in_rank_values = reinterpret_cast<const bool*>(&recv_is_token_in_rank_uint64);
                if (lane_id == num_topk_ranks)
                    src_meta = SourceMeta(rdma_rank, recv_is_token_in_rank_values);
                dst_send_buffers[num_topk_ranks ++] = reinterpret_cast<uint8_t*>(broadcast(send_buffer, i)) + slot_idx * num_bytes_per_rdma_token;
            }
            EP_DEVICE_ASSERT(num_topk_ranks <= kNumTopkRDMARanks);

            // Copy `x` into symmetric send buffer
            auto st_broadcast = [=](const int key, const int4& value) {
                #pragma unroll
                for (int j = 0; j < num_topk_ranks; ++ j)
                    st_na_global(reinterpret_cast<int4*>(dst_send_buffers[j]) + key, value);
            };
            UNROLLED_WARP_COPY(5, lane_id, hidden_int4, 0, x + token_idx * hidden_int4, ld_nc_global, st_broadcast);
            #pragma unroll
            for (int i = 0; i < num_topk_ranks; ++ i)
                dst_send_buffers[i] = reinterpret_cast<int4*>(dst_send_buffers[i]) + hidden_int4;

            // Copy source metadata into symmetric send buffer
            if (lane_id < num_topk_ranks)
                st_na_global(reinterpret_cast<SourceMeta*>(dst_send_buffers[lane_id]), src_meta);
            #pragma unroll
            for (int i = 0; i < num_topk_ranks; ++ i)
                dst_send_buffers[i] = reinterpret_cast<SourceMeta*>(dst_send_buffers[i]) + 1;

            // Copy `x_scales` into symmetric send buffer
            #pragma unroll
            for (int i = lane_id; i < num_scales; i += 32) {
                auto offset = token_idx * scale_token_stride + i * scale_hidden_stride;
                auto value = ld_nc_global(x_scales + offset);
                #pragma unroll
                for (int j = 0; j < num_topk_ranks; ++ j)
                    st_na_global(reinterpret_cast<float*>(dst_send_buffers[j]) + i, value);
            }
            #pragma unroll
            for (int i = 0; i < num_topk_ranks; ++ i)
                dst_send_buffers[i] = reinterpret_cast<float*>(dst_send_buffers[i]) + num_scales;

            // Copy `topk_idx` and `topk_weights` into symmetric send buffer
            #pragma unroll
            for (int i = lane_id; i < num_topk * num_topk_ranks; i += 32) {
                auto rank_idx = i / num_topk, copy_idx = i % num_topk;
                auto idx_value = static_cast<int>(ld_nc_global(topk_idx + token_idx * num_topk + copy_idx));
                auto weight_value = ld_nc_global(topk_weights + token_idx * num_topk + copy_idx);
                st_na_global(reinterpret_cast<int*>(dst_send_buffers[rank_idx]) + copy_idx, idx_value);
                st_na_global(reinterpret_cast<float*>(dst_send_buffers[rank_idx]) + num_topk + copy_idx, weight_value);
            }
            __syncwarp();

            // Release the transaction in the window
            if (is_token_in_rank_uint64 != 0) {
                // Acquire lock first
                acquire_lock(rdma_send_channel_lock + lane_id);
                auto latest_tail = rdma_send_channel_tail[lane_id];
                auto offset = rdma_tail_idx - latest_tail;
                while (offset >= 32) {
                    release_lock(rdma_send_channel_lock + lane_id);
                    acquire_lock(rdma_send_channel_lock + lane_id);
                    latest_tail = rdma_send_channel_tail[lane_id];
                    offset = rdma_tail_idx - latest_tail;
                }

                // Release the transaction slot
                // Add the bit and move the ones if possible
                auto window = rdma_send_channel_window[lane_id] | (1u << offset);
                if (offset == 0) {
                    auto num_empty_slots = (~window) == 0 ? 32 : __ffs(~window) - 1;
                    st_release_cta(rdma_send_channel_tail + lane_id, latest_tail + num_empty_slots);
                    window >>= num_empty_slots;
                }
                rdma_send_channel_window[lane_id] = window;

                // Release lock
                release_lock(rdma_send_channel_lock + lane_id);
            }
            __syncwarp();
        }
    } else if (warp_role == WarpRole::kRDMASenderCoordinator) {
        // NOTES: in case of splitting, the issued put at the end of the buffer
        EP_DEVICE_ASSERT(num_max_rdma_chunked_recv_tokens % num_max_rdma_chunked_send_tokens == 0);

        // Clean shared memory
        EP_STATIC_ASSERT(kNumRDMARanks <= 32, "Invalid number of RDMA ranks");
        (lane_id < kNumRDMARanks) ? (rdma_send_channel_lock[lane_id] = 0) : 0;
        (lane_id < kNumRDMARanks) ? (rdma_send_channel_tail[lane_id] = 0) : 0;
        (lane_id < kNumRDMARanks) ? (rdma_send_channel_window[lane_id] = 0) : 0;

        // Synchronize shared memory
        sync_rdma_sender_smem();

        // Get number of tokens to send for each RDMA rank
        int num_tokens_to_send = 0;
        if (lane_id < kNumRDMARanks) {
            num_tokens_to_send = rdma_channel_prefix_matrix[lane_id * num_channels + channel_id];
            if (channel_id > 0)
                num_tokens_to_send -= rdma_channel_prefix_matrix[lane_id * num_channels + channel_id - 1];
        }

        // Iterate all RDMA ranks
        int last_issued_tail = 0;
        auto start_time = clock64();
        while (__any_sync(0xffffffff, num_tokens_to_send > 0)) {
            // Timeout check
            if (clock64() - start_time > NUM_TIMEOUT_CYCLES and lane_id < kNumRDMARanks) {
                printf("DeepEP RDMA sender coordinator timeout, channel: %d, IB: %d, nvl %d, dst IB: %d, tail: %d, remaining: %d\n",
                       channel_id, rdma_rank, nvl_rank, lane_id, last_issued_tail, num_tokens_to_send);
                trap();
            }

            // TODO: try thread-level `put_nbi`?
            for (int i = 0, synced_num_tokens_to_send; i < kNumRDMARanks; ++ i) {
                // To mitigate incast congestion, shuffle the starting index of target rank for different ranks and channels
                int dst_rdma_rank = (i + channel_id + rdma_rank) % kNumRDMARanks;
                synced_num_tokens_to_send = __shfl_sync(0xffffffff, num_tokens_to_send, dst_rdma_rank);
                if (synced_num_tokens_to_send == 0)
                    continue;

                // Read the latest progress
                // NOTES: `rdma_send_channel_tail` does not need to be protected by lock
                auto processed_tail = __shfl_sync(0xffffffff, ld_acquire_cta(const_cast<const int*>(rdma_send_channel_tail + dst_rdma_rank)), 0);
                auto synced_last_issued_tail = __shfl_sync(0xffffffff, last_issued_tail, dst_rdma_rank);
                auto num_tokens_processed = processed_tail - synced_last_issued_tail;
                if (num_tokens_processed != synced_num_tokens_to_send and num_tokens_processed < num_max_rdma_chunked_send_tokens)
                    continue;

                // Issue RDMA send
                auto num_tokens_to_issue = min(num_tokens_processed, num_max_rdma_chunked_send_tokens);
                EP_DEVICE_ASSERT(num_tokens_to_issue >= 0 and num_tokens_to_issue <= synced_num_tokens_to_send);
                if (dst_rdma_rank != rdma_rank) {
                    auto dst_slot_idx = synced_last_issued_tail % num_max_rdma_chunked_recv_tokens;
                    EP_DEVICE_ASSERT(dst_slot_idx + num_tokens_to_issue <= num_max_rdma_chunked_recv_tokens);
                    const size_t num_bytes_per_msg = num_bytes_per_rdma_token * num_tokens_to_issue;
                    const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_channel_data.recv_buffer(rdma_rank) + dst_slot_idx * num_bytes_per_rdma_token);
                    const auto src_ptr = reinterpret_cast<uint64_t>(rdma_channel_data.send_buffer(dst_rdma_rank) + dst_slot_idx * num_bytes_per_rdma_token);
                    nvshmemi_ibgda_put_nbi_warp<true>(dst_ptr, src_ptr, num_bytes_per_msg,
                                                      translate_dst_rdma_rank<kLowLatencyMode>(dst_rdma_rank, nvl_rank), channel_id, lane_id, 0);
                } else {
                    // Lighter fence for local RDMA rank
                    memory_fence();
                }
                __syncwarp();

                // Update tails
                if (lane_id == dst_rdma_rank) {
                    last_issued_tail += num_tokens_to_issue;
                    num_tokens_to_send -= num_tokens_to_issue;
                    nvshmemi_ibgda_amo_nonfetch_add(rdma_channel_tail.buffer(rdma_rank), num_tokens_to_issue,
                                                    translate_dst_rdma_rank<kLowLatencyMode>(dst_rdma_rank, nvl_rank), channel_id, dst_rdma_rank == rdma_rank);
                }
                __syncwarp();
            }
        }
    } else if (warp_role == WarpRole::kRDMAAndNVLForwarder) {
        // RDMA consumers and NVL producers
        const auto dst_nvl_rank = target_rank;
        const auto dst_rank = rdma_rank * NUM_MAX_NVL_PEERS + dst_nvl_rank;
        const auto dst_rank_expert_begin = dst_rank * (num_experts / num_ranks);
        const auto dst_rank_expert_end = dst_rank_expert_begin + (num_experts / num_ranks);

        // Wait counters to arrive
        int num_tokens_to_recv_from_rdma = 0, src_rdma_channel_prefix = 0;
        EP_DEVICE_ASSERT(kNumRDMARanks <= 32);
        auto start_time = clock64();
        if (lane_id < kNumRDMARanks) {
            while (true) {
                auto meta_0 = ld_volatile_global(rdma_channel_meta.recv_buffer(lane_id) + dst_nvl_rank);
                auto meta_1 = ld_volatile_global(rdma_channel_meta.recv_buffer(lane_id) + NUM_MAX_NVL_PEERS + dst_nvl_rank);
                auto meta_2 = ld_volatile_global(rdma_channel_meta.recv_buffer(lane_id) + NUM_MAX_NVL_PEERS * 2);
                auto meta_3 = ld_volatile_global(rdma_channel_meta.recv_buffer(lane_id) + NUM_MAX_NVL_PEERS * 2 + 1);
                if (meta_0 < 0 and meta_1 < 0 and meta_2 < 0 and meta_3 < 0) {
                    // Notify NVL ranks
                    int start_sum = -meta_0 - 1, end_sum = -meta_1 - 1;
                    EP_DEVICE_ASSERT(start_sum >= 0 and end_sum >= 0 and end_sum >= start_sum);
                    st_relaxed_sys_global(nvl_channel_prefix_start.buffer() + lane_id, -start_sum - 1);
                    st_relaxed_sys_global(nvl_channel_prefix_end.buffer() + lane_id, -end_sum - 1);

                    // Save RDMA channel received token count
                    src_rdma_channel_prefix = -meta_2 - 1;
                    auto src_rdma_channel_prefix_1 = -meta_3 - 1;
                    num_tokens_to_recv_from_rdma = src_rdma_channel_prefix_1 - src_rdma_channel_prefix;
                    if (not kCachedMode)
                        recv_rdma_channel_prefix_matrix[lane_id * num_channels + channel_id] = src_rdma_channel_prefix_1;
                    src_rdma_channel_prefix += lane_id == 0 ? 0 : recv_rdma_rank_prefix_sum[lane_id - 1];
                    EP_DEVICE_ASSERT(num_tokens_to_recv_from_rdma >= 0);
                    break;
                }

                // Timeout check
                if (clock64() - start_time > NUM_TIMEOUT_CYCLES) {
                    printf("DeepEP dispatch forwarder timeout (RDMA meta), channel: %d, RDMA: %d, nvl: %d, src RDMA lane: %d, dst NVL: %d, meta: %d, %d, %d, %d\n",
                           channel_id, rdma_rank, nvl_rank, lane_id, dst_nvl_rank, meta_0, meta_1, meta_2, meta_3);
                    trap();
                }
            }
        }
        __syncwarp();

        // Shift cached head
        send_nvl_head += src_rdma_channel_prefix * NUM_MAX_NVL_PEERS + dst_nvl_rank;

        // Wait shared memory to be cleaned
        sync_forwarder_smem();

        // Forward tokens from RDMA buffer
        // NOTES: always start from the local rank
        int src_rdma_rank = sm_id % kNumRDMARanks;
        int cached_rdma_channel_head = 0, cached_rdma_channel_tail = 0;
        int cached_nvl_channel_head = 0, cached_nvl_channel_tail = 0, rdma_nvl_token_idx = 0;
        while (__any_sync(0xffffffff, num_tokens_to_recv_from_rdma > 0)) {
            // Check destination queue emptiness, or wait a buffer to be released
            start_time = clock64();
            while (lane_id == 0) {
                int num_used_slots = cached_nvl_channel_tail - cached_nvl_channel_head;
                if (num_max_nvl_chunked_recv_tokens - num_used_slots >= num_max_nvl_chunked_send_tokens)
                    break;
                cached_nvl_channel_head = ld_volatile_global(nvl_channel_head.buffer());

                // Timeout check
                if (clock64() - start_time > NUM_TIMEOUT_CYCLES) {
                    printf("DeepEP dispatch forwarder timeout (NVL check), channel: %d, RDMA: %d, nvl: %d, dst NVL: %d, head: %d, tail: %d\n",
                           channel_id, rdma_rank, nvl_rank, dst_nvl_rank, ld_volatile_global(nvl_channel_head.buffer()), cached_nvl_channel_tail);
                    trap();
                }
            }
            __syncwarp();

            // Find next source RDMA rank (round-robin)
            start_time = clock64();
            while (true) {
                src_rdma_rank = (src_rdma_rank + 1) % kNumRDMARanks;
                if (__shfl_sync(0xffffffff, num_tokens_to_recv_from_rdma, src_rdma_rank) > 0) {
                    if (lane_id == src_rdma_rank and cached_rdma_channel_head == cached_rdma_channel_tail)
                        cached_rdma_channel_tail = static_cast<int>(ld_acquire_sys_global(rdma_channel_tail.buffer(src_rdma_rank)));
                    if (__shfl_sync(0xffffffff, cached_rdma_channel_tail > cached_rdma_channel_head, src_rdma_rank))
                        break;
                }

                // Timeout check
                if (clock64() - start_time > NUM_TIMEOUT_CYCLES and lane_id < kNumRDMARanks) {
                    printf("DeepEP dispatch forwarder timeout (RDMA check), channel: %d, RDMA: %d, nvl: %d, dst NVL: %d, src RDMA lane: %d, head: %d, tail: %d, expected: %d\n",
                           channel_id, rdma_rank, nvl_rank, dst_nvl_rank, lane_id, cached_rdma_channel_head, cached_rdma_channel_tail, num_tokens_to_recv_from_rdma);
                    trap();
                }
            }
            auto src_rdma_head = __shfl_sync(0xffffffff, cached_rdma_channel_head, src_rdma_rank);
            auto src_rdma_tail = __shfl_sync(0xffffffff, cached_rdma_channel_tail, src_rdma_rank);

            // Iterate over every token from the RDMA buffer
            for (int i = src_rdma_head, num_tokens_sent = 0; i < src_rdma_tail; ++ i) {
                auto rdma_slot_idx = i % num_max_rdma_chunked_recv_tokens;
                void* shifted = rdma_channel_data.recv_buffer(src_rdma_rank) + rdma_slot_idx * num_bytes_per_rdma_token;
                auto src_meta = ld_nc_global(reinterpret_cast<SourceMeta*>(static_cast<int8_t*>(shifted) + hidden_bytes));
                lane_id == src_rdma_rank ? (num_tokens_to_recv_from_rdma -= 1) : 0;
                bool is_in_dst_nvl_rank = src_meta.is_token_in_nvl_rank(dst_nvl_rank);
                if (lane_id == src_rdma_rank) {
                    auto cached_head = is_in_dst_nvl_rank ? rdma_nvl_token_idx : -1;
                    rdma_nvl_token_idx += is_in_dst_nvl_rank;
                    if (not kCachedMode)
                        send_nvl_head[i * NUM_MAX_NVL_PEERS] = cached_head;
                }
                if (not is_in_dst_nvl_rank)
                    continue;

                // Get an empty slot
                int dst_slot_idx = (cached_nvl_channel_tail ++) % num_max_nvl_chunked_recv_tokens;

                // Copy data
                UNROLLED_WARP_COPY(5, lane_id, hidden_int4,
                                   nvl_channel_x.buffer() + dst_slot_idx * hidden_int4,
                                   reinterpret_cast<int4*>(shifted),
                                   ld_nc_global, st_na_global);
                shifted = static_cast<int4*>(shifted) + hidden_int4;

                // Copy source meta
                if (lane_id == 0)
                    st_na_global(nvl_channel_src_meta.buffer() + dst_slot_idx, src_meta);
                shifted = static_cast<SourceMeta*>(shifted) + 1;

                // Copy `x_scales`
                UNROLLED_WARP_COPY(1, lane_id, num_scales,
                                   nvl_channel_x_scales.buffer() + dst_slot_idx * num_scales,
                                   reinterpret_cast<float*>(shifted),
                                   ld_nc_global, st_na_global);
                shifted = static_cast<float*>(shifted) + num_scales;

                // Copy `topk_idx` and `topk_weights`
                // NOTES: do not use `shifted` after this `if`, because only several lanes are shifted
                if (lane_id < num_topk) {
                    // Read
                    auto idx_value = ld_nc_global(static_cast<int*>(shifted) + lane_id);
                    shifted = static_cast<int*>(shifted) + num_topk;
                    auto weight_value = ld_nc_global(static_cast<float*>(shifted) + lane_id);

                    // Transform and write
                    idx_value = (idx_value >= dst_rank_expert_begin and idx_value < dst_rank_expert_end) ? idx_value - dst_rank_expert_begin : -1;
                    st_na_global(nvl_channel_topk_idx.buffer() + dst_slot_idx * num_topk + lane_id, idx_value);
                    weight_value = idx_value >= 0 ? weight_value : 0.0f;
                    st_na_global(nvl_channel_topk_weights.buffer() + dst_slot_idx * num_topk + lane_id, weight_value);
                }

                // In case of insufficient NVL buffers, early stopping
                if ((++ num_tokens_sent) == num_max_nvl_chunked_send_tokens)
                    src_rdma_tail = i + 1;
            }

            // Sync head index
            if (lane_id == src_rdma_rank)
                forward_channel_head[dst_nvl_rank][src_rdma_rank] = (cached_rdma_channel_head = src_rdma_tail);

            // Move tail index
            __syncwarp();
            if (lane_id == 0)
                st_release_sys_global(nvl_channel_tail.buffer(), cached_nvl_channel_tail);
        }

        // Retired
        __syncwarp();
        if (lane_id == 0)
            forward_channel_retired[dst_nvl_rank] = true;
    } else if (warp_role == WarpRole::kForwarderCoordinator) {
        // Extra warps for forwarder coordinator should exit directly
        if (target_rank > 0)
            return;

        // Forward warp coordinator
        EP_STATIC_ASSERT(kNumRDMARanks <= 32, "Invalid number of RDMA peers");

        // Clean shared memory
        EP_STATIC_ASSERT(NUM_MAX_NVL_PEERS <= 32, "Invalid number of NVL peers");
        #pragma unroll
        for (int i = lane_id; i < kNumRDMARanks * NUM_MAX_NVL_PEERS; i += 32)
            forward_channel_head[i % NUM_MAX_NVL_PEERS][i / NUM_MAX_NVL_PEERS] = 0;
        if (lane_id < NUM_MAX_NVL_PEERS)
            forward_channel_retired[lane_id] = false;
        sync_forwarder_smem();

        int last_head = 0, target_rdma = lane_id < kNumRDMARanks ? lane_id : 0;
        while (true) {
            // Find minimum head
            int min_head = std::numeric_limits<int>::max();
            #pragma unroll
            for (int i = 0; i < NUM_MAX_NVL_PEERS; ++ i) if (not forward_channel_retired[i])
                min_head = min(min_head, forward_channel_head[i][target_rdma]);
            if (__all_sync(0xffffffff, min_head == std::numeric_limits<int>::max()))
                break;

            // Update remote head
            if (min_head != std::numeric_limits<int>::max() and min_head >= last_head + num_max_rdma_chunked_send_tokens and lane_id < kNumRDMARanks) {
                nvshmemi_ibgda_amo_nonfetch_add(rdma_channel_head.buffer(rdma_rank), min_head - last_head,
                                                translate_dst_rdma_rank<kLowLatencyMode>(lane_id, nvl_rank), channel_id + num_channels, lane_id == rdma_rank);
                last_head = min_head;
            }

            // Nanosleep and let other warps work
            __nanosleep(NUM_WAIT_NANOSECONDS);
        }
    } else {
        // NVL consumers
        // Retrieve rank offset from barrier results (each lane's register stores an RDMA rank)
        int src_nvl_rank = target_rank, total_offset = 0;
        EP_STATIC_ASSERT(kNumRDMARanks <= 32, "Invalid number of RDMA peers");
        if (lane_id < kNumRDMARanks and lane_id * NUM_MAX_NVL_PEERS + src_nvl_rank > 0)
            total_offset = recv_gbl_rank_prefix_sum[lane_id * NUM_MAX_NVL_PEERS + src_nvl_rank - 1];

        // Receive channel offsets
        int start_offset = 0, end_offset = 0, num_tokens_to_recv;
        auto start_time = clock64();
        while (lane_id < kNumRDMARanks) {
            start_offset = ld_volatile_global(nvl_channel_prefix_start.buffer() + lane_id);
            end_offset = ld_volatile_global(nvl_channel_prefix_end.buffer() + lane_id);
            if (start_offset < 0 and end_offset < 0) {
                start_offset = -start_offset - 1, end_offset = -end_offset - 1;
                total_offset += start_offset;
                break;
            }

            // Timeout check
            if (clock64() - start_time > NUM_TIMEOUT_CYCLES) {
                printf("DeepEP dispatch NVL receiver timeout, channel: %d, RDMA: %d, nvl: %d, src RDMA: %d, src nvl: %d, start: %d, end: %d\n",
                       channel_id, rdma_rank, nvl_rank, lane_id, src_nvl_rank, start_offset, end_offset);
                trap();
            }
        }
        num_tokens_to_recv = warp_reduce_sum(end_offset - start_offset);

        // Save for combine usage
        if (lane_id < kNumRDMARanks and not kCachedMode)
            recv_gbl_channel_prefix_matrix[(lane_id * NUM_MAX_NVL_PEERS + src_nvl_rank) * num_channels + channel_id] = total_offset;
        __syncwarp();

        int cached_channel_head_idx = 0, cached_channel_tail_idx = 0;
        while (num_tokens_to_recv > 0) {
            // Check channel status by lane 0
            start_time = clock64();
            while (lane_id == 0) {
                // Ready to copy
                if (cached_channel_head_idx != cached_channel_tail_idx)
                    break;
                cached_channel_tail_idx = ld_acquire_sys_global(nvl_channel_tail.buffer());

                // Timeout check
                if (clock64() - start_time > NUM_TIMEOUT_CYCLES) {
                    printf("DeepEP dispatch NVL receiver timeout, channel: %d, RDMA: %d, nvl: %d, src NVL: %d, head: %d, tail: %d\n",
                           channel_id, rdma_rank, nvl_rank, src_nvl_rank, cached_channel_head_idx, cached_channel_tail_idx);
                    trap();
                }
            }

            // Sync queue tail
            cached_channel_tail_idx = __shfl_sync(0xffffffff, cached_channel_tail_idx, 0);

            // Copy data
            int num_recv_tokens = cached_channel_tail_idx - cached_channel_head_idx;
            for (int chunk_idx = 0; chunk_idx < num_recv_tokens; ++ chunk_idx, -- num_tokens_to_recv) {
                int token_idx_in_buffer = (cached_channel_head_idx ++) % num_max_nvl_chunked_recv_tokens;
                auto meta = ld_nc_global(nvl_channel_src_meta.buffer() + token_idx_in_buffer);
                int64_t recv_token_idx = __shfl_sync(0xffffffff, total_offset, meta.src_rdma_rank);
                (lane_id == meta.src_rdma_rank) ? (total_offset += 1) : 0;

                // Copy data
                UNROLLED_WARP_COPY(5, lane_id, hidden_int4,
                                   recv_x + recv_token_idx * hidden_int4,
                                   nvl_channel_x.buffer() + token_idx_in_buffer * hidden_int4,
                                   ld_nc_global, st_na_global);

                // Copy source meta
                if (lane_id == 0 and not kCachedMode)
                    st_na_global(recv_src_meta + recv_token_idx, meta);

                // Copy scales
                UNROLLED_WARP_COPY(1, lane_id, num_scales,
                                   recv_x_scales + recv_token_idx * num_scales,
                                   nvl_channel_x_scales.buffer() + token_idx_in_buffer * num_scales,
                                   ld_nc_global, st_na_global);

                // Copy `topk_idx` and `topk_weights`
                if (lane_id < num_topk) {
                    auto recv_idx = recv_token_idx * num_topk + lane_id;
                    auto buffer_idx = token_idx_in_buffer * num_topk + lane_id;
                    int local_recv_topk_idx = static_cast<int>(ld_nc_global(nvl_channel_topk_idx.buffer() + buffer_idx));
                    local_recv_topk_idx = local_recv_topk_idx == -1 ? num_experts : local_recv_topk_idx + global_expert_id_offset;
                    st_na_global(recv_topk_idx + recv_idx, local_recv_topk_idx);
                    st_na_global(recv_topk_weights + recv_idx, ld_nc_global(nvl_channel_topk_weights.buffer() + buffer_idx));
                }
            }

            // Move queue
            __syncwarp();
            if (lane_id == 0)
                st_relaxed_sys_global(nvl_channel_head.buffer(), cached_channel_head_idx);
        }
    }
}

void dispatch(void* recv_x, float* recv_x_scales, int* recv_topk_idx, float* recv_topk_weights, void* recv_src_meta,
              const void* x, const float* x_scales, const int* topk_idx, const float* topk_weights,
              int* send_rdma_head, int* send_nvl_head,
              int* recv_rdma_channel_prefix_matrix, int* recv_gbl_channel_prefix_matrix,
              const int* rdma_channel_prefix_matrix, const int* recv_rdma_rank_prefix_sum,
              const int* gbl_channel_prefix_matrix, const int* recv_gbl_rank_prefix_sum,
              const bool* is_token_in_rank,
              int num_tokens, int hidden_int4, int num_scales, int num_topk, int num_experts, int global_expert_id_offset,
              int scale_token_stride, int scale_hidden_stride,
              void* rdma_buffer_ptr, int num_max_rdma_chunked_send_tokens, int num_max_rdma_chunked_recv_tokens,
              void** buffer_ptrs, int num_max_nvl_chunked_send_tokens, int num_max_nvl_chunked_recv_tokens,
              int rank, int num_ranks, bool is_cached_dispatch,
              hipStream_t stream, int num_channels, bool low_latency_mode) {
    constexpr int kNumDispatchRDMASenderWarps = 7;

    // Make sure never OOB
    EP_HOST_ASSERT(static_cast<int64_t>(num_scales) * scale_hidden_stride < std::numeric_limits<int>::max());

#define DISPATCH_LAUNCH_CASE(num_rdma_ranks) { \
    auto dispatch_func = low_latency_mode ? \
        (is_cached_dispatch ? dispatch<true, num_rdma_ranks, true, kNumDispatchRDMASenderWarps> : dispatch<true, num_rdma_ranks, false, kNumDispatchRDMASenderWarps>) : \
        (is_cached_dispatch ? dispatch<false, num_rdma_ranks, true, kNumDispatchRDMASenderWarps> : dispatch<false, num_rdma_ranks, false, kNumDispatchRDMASenderWarps>); \
    LAUNCH_KERNEL(&cfg, dispatch_func, \
                  reinterpret_cast<int4*>(recv_x), recv_x_scales, recv_topk_idx, recv_topk_weights, reinterpret_cast<SourceMeta*>(recv_src_meta), \
                  reinterpret_cast<const int4*>(x), x_scales, topk_idx, topk_weights, \
                  send_rdma_head, send_nvl_head, \
                  recv_rdma_channel_prefix_matrix, recv_gbl_channel_prefix_matrix, \
                  rdma_channel_prefix_matrix, recv_rdma_rank_prefix_sum, \
                  gbl_channel_prefix_matrix, recv_gbl_rank_prefix_sum, \
                  is_token_in_rank, \
                  num_tokens, hidden_int4, num_scales, num_topk, num_experts, global_expert_id_offset, \
                  scale_token_stride, scale_hidden_stride, \
                  rdma_buffer_ptr, num_max_rdma_chunked_send_tokens, num_max_rdma_chunked_recv_tokens, \
                  buffer_ptrs, num_max_nvl_chunked_send_tokens, num_max_nvl_chunked_recv_tokens, \
                  rank, num_ranks); } break

    EP_HOST_ASSERT((topk_idx == nullptr)  == (topk_weights == nullptr));
    EP_HOST_ASSERT((recv_topk_idx == nullptr) == (recv_topk_weights == nullptr));

    SETUP_LAUNCH_CONFIG(num_channels * 2, (kNumDispatchRDMASenderWarps + 1 + NUM_MAX_NVL_PEERS) * 32, stream);
    SWITCH_RDMA_RANKS(DISPATCH_LAUNCH_CASE);
#undef DISPATCH_LAUNCH_CASE
}

template <bool kLowLatencyMode>
__global__ void cached_notify(const int rdma_clean_offset, const int rdma_num_int_clean,
                              const int nvl_clean_offset, const int nvl_num_int_clean,
                              int* combined_rdma_head, int num_combined_tokens, int num_channels,
                              const int* rdma_channel_prefix_matrix, const int* rdma_rank_prefix_sum, int* combined_nvl_head,
                              void* rdma_buffer_ptr,
                              void** buffer_ptrs, int** barrier_signal_ptrs, int rank, int num_ranks,
                              bool is_cached_dispatch, const nvshmem_team_t rdma_team) {
    auto sm_id = static_cast<int>(blockIdx.x);
    auto thread_id = static_cast<int>(threadIdx.x);
    auto num_threads = static_cast<int>(blockDim.x);
    auto num_warps = num_threads / 32;
    auto warp_id = thread_id / 32;
    auto lane_id = get_lane_id();

    auto nvl_rank = rank % NUM_MAX_NVL_PEERS;
    auto num_rdma_ranks = num_ranks / NUM_MAX_NVL_PEERS;

    // Using two SMs, which clean the RDMA/NVL buffer respectively
    if (sm_id == 0) {
        // Barrier for RDMA
        if (thread_id == 0)
            nvshmem_sync_with_same_gpu_idx<kLowLatencyMode>(rdma_team);
        __syncthreads();

        // Clean
        auto rdma_buffer_ptr_int = static_cast<int*>(rdma_buffer_ptr);
        #pragma unroll
        for (int i = thread_id; i < rdma_num_int_clean; i += num_threads)
            rdma_buffer_ptr_int[rdma_clean_offset + i] = 0;
        __syncthreads();

        // Barrier again
        if (thread_id == 0)
            nvshmem_sync_with_same_gpu_idx<kLowLatencyMode>(rdma_team);
    } else if (sm_id == 1) {
        // Barrier for NVL
        barrier_block<NUM_MAX_NVL_PEERS, true>(barrier_signal_ptrs, nvl_rank);

        // Clean
        auto nvl_buffer_ptr_int = static_cast<int*>(buffer_ptrs[nvl_rank]);
        #pragma unroll
        for (int i = thread_id; i < nvl_num_int_clean; i += num_threads)
            nvl_buffer_ptr_int[nvl_clean_offset + i] = 0;

        // Barrier again
        barrier_block<NUM_MAX_NVL_PEERS>(barrier_signal_ptrs, nvl_rank);
    } else if (sm_id == 2) {
        if (is_cached_dispatch)
            return;

        EP_DEVICE_ASSERT(num_warps >= num_channels);
        EP_DEVICE_ASSERT(num_rdma_ranks <= 32);

        // Iterate in reverse order
        if (lane_id < num_rdma_ranks and warp_id < num_channels) {
            int token_start_idx, token_end_idx;
            get_channel_task_range(num_combined_tokens, num_channels, warp_id, token_start_idx, token_end_idx);

            // NOTES: `1 << 25` is a heuristic large number
            int last_head = 1 << 25;
            for (int token_idx = token_end_idx - 1; token_idx >= token_start_idx; -- token_idx) {
                auto current_head = __ldg(combined_rdma_head + token_idx * num_rdma_ranks + lane_id);
                if (current_head < 0) {
                    combined_rdma_head[token_idx * num_rdma_ranks + lane_id] = -last_head - 1;
                } else {
                    last_head = current_head;
                }
            }
        }
    } else {
        if (is_cached_dispatch)
            return;

        EP_DEVICE_ASSERT(num_warps >= num_channels);
        EP_DEVICE_ASSERT(rdma_channel_prefix_matrix != nullptr and rdma_rank_prefix_sum != nullptr);
        EP_STATIC_ASSERT(NUM_MAX_NVL_PEERS <= 32, "Too many NVL peers");

        if (lane_id < NUM_MAX_NVL_PEERS and warp_id < num_channels) {
            for (int dst_rdma_rank = sm_id - 3; dst_rdma_rank < num_rdma_ranks; dst_rdma_rank += num_channels * 2 - 3) {
                // Iterate in reverse order
                int token_start_idx = warp_id == 0 ? 0 : rdma_channel_prefix_matrix[dst_rdma_rank * num_channels + warp_id - 1];
                int token_end_idx = rdma_channel_prefix_matrix[dst_rdma_rank * num_channels + warp_id];
                int shift = dst_rdma_rank == 0 ? 0 : rdma_rank_prefix_sum[dst_rdma_rank - 1];
                token_start_idx += shift, token_end_idx += shift;

                // NOTES: `1 << 25` is a heuristic large number
                int last_head = 1 << 25;
                #pragma unroll
                for (int token_idx = token_end_idx - 1; token_idx >= token_start_idx; -- token_idx)  {
                    auto current_head = __ldg(combined_nvl_head + token_idx * NUM_MAX_NVL_PEERS + lane_id);
                    if (current_head < 0) {
                        combined_nvl_head[token_idx * NUM_MAX_NVL_PEERS + lane_id] = -last_head - 1;
                    } else {
                        last_head = current_head;
                    }
                }
            }
        }
    }
}

void cached_notify(int hidden_int4, int num_scales, int num_topk_idx, int num_topk_weights,
                   int num_ranks, int num_channels, int num_combined_tokens, int* combined_rdma_head,
                   const int* rdma_channel_prefix_matrix, const int* rdma_rank_prefix_sum, int* combined_nvl_head,
                   void* rdma_buffer_ptr, int num_max_rdma_chunked_recv_tokens,
                   void** buffer_ptrs, int num_max_nvl_chunked_recv_tokens,
                   int** barrier_signal_ptrs, int rank, hipStream_t stream,
                   int64_t num_rdma_bytes, int64_t num_nvl_bytes,
                   bool is_cached_dispatch, bool low_latency_mode) {
    const int num_threads = std::max(128, 32 * num_channels);
    const auto num_rdma_ranks = num_ranks / NUM_MAX_NVL_PEERS;

    // Get clean meta
    auto rdma_clean_meta = get_rdma_clean_meta(hidden_int4, num_scales, num_topk_idx, num_topk_weights, num_rdma_ranks, num_max_rdma_chunked_recv_tokens, num_channels);
    auto nvl_clean_meta = get_nvl_clean_meta(hidden_int4, num_scales, num_topk_idx, num_topk_weights, num_rdma_ranks, NUM_MAX_NVL_PEERS, num_max_nvl_chunked_recv_tokens, num_channels);
    EP_HOST_ASSERT((rdma_clean_meta.first + rdma_clean_meta.second) * sizeof(int) <= num_rdma_bytes);
    EP_HOST_ASSERT((nvl_clean_meta.first + nvl_clean_meta.second) * sizeof(int) <= num_nvl_bytes);
    EP_HOST_ASSERT(num_rdma_bytes < std::numeric_limits<int>::max());
    EP_HOST_ASSERT(num_nvl_bytes < std::numeric_limits<int>::max());
    EP_HOST_ASSERT(num_channels * 2 > 3);

    // Launch kernel
    auto cached_notify_func = low_latency_mode ? cached_notify<true> : cached_notify<false>;
    SETUP_LAUNCH_CONFIG(num_channels * 2, num_threads, stream);
    LAUNCH_KERNEL(&cfg, cached_notify_func,
                  rdma_clean_meta.first, rdma_clean_meta.second,
                  nvl_clean_meta.first, nvl_clean_meta.second,
                  combined_rdma_head, num_combined_tokens, num_channels,
                  rdma_channel_prefix_matrix, rdma_rank_prefix_sum, combined_nvl_head,
                  rdma_buffer_ptr,
                  buffer_ptrs, barrier_signal_ptrs, rank, num_ranks,
                  is_cached_dispatch, cpu_rdma_team);
}

template <int kNumRanks, bool kMaybeWithBias, typename dtype_t, int kMaxNumRanks, typename ReceiveFn, typename ReceiveTWFn>
__device__ int combine_token(bool is_token_in_rank, int head_idx,
                             int lane_id, int hidden_int4, int num_topk,
                             int4* combined_row, float* combined_topk_weights,
                             const int4* bias_0_int4, const int4* bias_1_int4,
                             int num_max_recv_tokens, const ReceiveFn& recv_fn, const ReceiveTWFn& recv_tw_fn) {
    constexpr auto kDtypePerInt4 = sizeof(int4) / sizeof(dtype_t);

    // Broadcast current heads
    // Lane `i` holds the head of rank `i` and `is_token_in_rank`
    EP_STATIC_ASSERT(kMaxNumRanks <= 32, "Too many ranks");
    int num_topk_ranks = 0, topk_ranks[kMaxNumRanks], slot_indices[kMaxNumRanks];
    #pragma unroll
    for (int i = 0; i < kNumRanks; ++ i) if (__shfl_sync(0xffffffff, is_token_in_rank, i)) {
        slot_indices[num_topk_ranks] = __shfl_sync(0xffffffff, head_idx, i) % num_max_recv_tokens;
        topk_ranks[num_topk_ranks ++] = i;
    }
    EP_DEVICE_ASSERT(num_topk_ranks <= kMaxNumRanks);

    // Reduce data
    #pragma unroll
    for (int i = lane_id; i < hidden_int4; i += 32) {
        // Read bias
        // TODO: make it as a finer-grained template
        int4 bias_0_value_int4, bias_1_value_int4;
        if (kMaybeWithBias) {
            bias_0_value_int4 = bias_0_int4 != nullptr ? ld_nc_global(bias_0_int4 + i) : make_int4(0, 0, 0, 0);
            bias_1_value_int4 = bias_1_int4 != nullptr ? ld_nc_global(bias_1_int4 + i) : make_int4(0, 0, 0, 0);
        }

        // Read buffers
        // TODO: maybe too many registers here
        int4 recv_value_int4[kMaxNumRanks];
        #pragma unroll
        for (int j = 0; j < num_topk_ranks; ++ j)
            recv_value_int4[j] = recv_fn(topk_ranks[j], slot_indices[j], i);
        
        // Clean
        // Reduce bias
        float values[kDtypePerInt4] = {0};
        if (kMaybeWithBias) {
            auto bias_0_values = reinterpret_cast<const dtype_t*>(&bias_0_value_int4);
            auto bias_1_values = reinterpret_cast<const dtype_t*>(&bias_1_value_int4);
            #pragma unroll
            for (int j = 0; j < kDtypePerInt4; ++ j)
                values[j] = static_cast<float>(bias_0_values[j]) + static_cast<float>(bias_1_values[j]);
        }

        // Reduce all-to-all results
        #pragma unroll
        for (int j = 0; j < num_topk_ranks; ++ j) {
            auto recv_value_dtypes = reinterpret_cast<const dtype_t*>(&recv_value_int4[j]);
            #pragma unroll
            for (int k = 0; k < kDtypePerInt4; ++ k)
                values[k] += static_cast<float>(recv_value_dtypes[k]);
        }

        // Cast back to `dtype_t` and write
        int4 out_int4;
        auto out_dtypes = reinterpret_cast<dtype_t*>(&out_int4);
        #pragma unroll
        for (int j = 0; j < kDtypePerInt4; ++ j)
            out_dtypes[j] = static_cast<dtype_t>(values[j]);
        st_na_global(combined_row + i, out_int4);
    }

    // Reduce `topk_weights`
    if (lane_id < num_topk) {
        float value = 0;
        #pragma unroll
        for (int i = 0; i < num_topk_ranks; ++ i)
            value += recv_tw_fn(topk_ranks[i], slot_indices[i], lane_id);
        st_na_global(combined_topk_weights + lane_id, value);
    }

    // Return the minimum top-k rank
    return topk_ranks[0];
}

template<bool kLowLatencyMode,
         int kNumRDMARanks, typename dtype_t,
         int kNumCombineForwarderWarps,
         int kNumTopkRDMARanks = get_num_topk_rdma_ranks(kNumRDMARanks),
         int kNumWarpsPerForwarder = (kNumCombineForwarderWarps / kNumRDMARanks > 0) ? kNumCombineForwarderWarps / kNumRDMARanks : 1,
         int kNumForwarders = kNumRDMARanks * kNumWarpsPerForwarder,
         int kNumRDMAReceivers = kNumForwarders + NUM_MAX_NVL_PEERS>
__global__ void __launch_bounds__((NUM_MAX_NVL_PEERS + 1 + kNumForwarders) * 32, 1)
combine(int4* combined_x, float* combined_topk_weights,
        const bool* is_combined_token_in_rank,
        const int4* x, const float* topk_weights,
        const int4* bias_0, const int4* bias_1,
        const int* combined_rdma_head, const int* combined_nvl_head,
        const SourceMeta* src_meta, const int* rdma_channel_prefix_matrix, const int* rdma_rank_prefix_sum, const int* gbl_channel_prefix_matrix,
        int num_tokens, int num_combined_tokens, int hidden, int num_topk,
        void* rdma_buffer_ptr, int num_max_rdma_chunked_send_tokens, int num_max_rdma_chunked_recv_tokens,
        void** buffer_ptrs, int num_max_nvl_chunked_send_tokens, int num_max_nvl_chunked_recv_tokens,
        int rank, int num_ranks) {
    enum class WarpRole {
        kNVLSender,
        kNVLAndRDMAForwarder,
        kRDMAReceiver,
        kCoordinator
    };

    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto num_threads = static_cast<int>(blockDim.x), num_warps = num_threads / 32;
    const auto thread_id = static_cast<int>(threadIdx.x), lane_id = get_lane_id();
    const auto num_channels = static_cast<int>(gridDim.x) / 2, channel_id = sm_id / 2;
    const bool is_rdma_receiver_sm = sm_id % 2 == 1;

    EP_DEVICE_ASSERT(num_topk <= 32);
    EP_DEVICE_ASSERT(hidden % (sizeof(int4) / sizeof(dtype_t)) == 0);
    const auto hidden_int4 = hidden / (sizeof(int4) / sizeof(dtype_t));

    // NOTES: we decouple a channel into 2 SMs
    const auto rdma_rank = rank / NUM_MAX_NVL_PEERS, nvl_rank = rank % NUM_MAX_NVL_PEERS;
    auto role_meta = [=]() -> std::pair<WarpRole, int> {
        auto warp_id = thread_id / 32;
        if (not is_rdma_receiver_sm) {
            if (warp_id < NUM_MAX_NVL_PEERS) {
                auto shuffled_warp_id = warp_id;
                shuffled_warp_id = (shuffled_warp_id + channel_id) % NUM_MAX_NVL_PEERS;
                return {WarpRole::kNVLSender, shuffled_warp_id};
            } else if (warp_id < NUM_MAX_NVL_PEERS + kNumForwarders) {
                auto shuffled_warp_id = warp_id - NUM_MAX_NVL_PEERS;
                shuffled_warp_id = (shuffled_warp_id + channel_id) % kNumForwarders;
                return {WarpRole::kNVLAndRDMAForwarder, shuffled_warp_id};
            } else {
                return {WarpRole::kCoordinator, 0};
            }
        } else {
            if (warp_id < NUM_MAX_NVL_PEERS + kNumForwarders) {
                return {WarpRole::kRDMAReceiver, warp_id};
            } else {
                return {WarpRole::kCoordinator, 0};
            }
        }
    }();
    auto warp_role = role_meta.first;
    auto warp_id = role_meta.second;

    EP_DEVICE_ASSERT(num_warps == NUM_MAX_NVL_PEERS + kNumForwarders + 1);
    auto num_max_nvl_chunked_recv_tokens_per_rdma = num_max_nvl_chunked_recv_tokens / kNumRDMARanks;

    if (warp_role == WarpRole::kNVLSender) {
        // NVL producers
        const auto dst_nvl_rank = warp_id;

        // NVL layouts
        // NOTES: to avoid deadlocks, we use separate NVL buffers for different RDMA sources
        auto dst_buffer_ptr = buffer_ptrs[dst_nvl_rank], local_buffer_ptr = buffer_ptrs[nvl_rank];
        auto nvl_channel_x = AsymBuffer<int4>(dst_buffer_ptr, num_max_nvl_chunked_recv_tokens * hidden_int4, NUM_MAX_NVL_PEERS, channel_id, num_channels, nvl_rank).advance_also(local_buffer_ptr);
        auto nvl_channel_src_meta = AsymBuffer<SourceMeta>(dst_buffer_ptr, num_max_nvl_chunked_recv_tokens, NUM_MAX_NVL_PEERS, channel_id, num_channels, nvl_rank).advance_also(local_buffer_ptr);
        auto nvl_channel_topk_weights = AsymBuffer<float>(dst_buffer_ptr, num_max_nvl_chunked_recv_tokens * num_topk, NUM_MAX_NVL_PEERS, channel_id, num_channels, nvl_rank).advance_also(local_buffer_ptr);
        auto nvl_channel_head = AsymBuffer<int>(local_buffer_ptr, kNumRDMARanks, NUM_MAX_NVL_PEERS, channel_id, num_channels, dst_nvl_rank).advance_also(dst_buffer_ptr);
        auto nvl_channel_tail = AsymBuffer<int>(dst_buffer_ptr, kNumRDMARanks, NUM_MAX_NVL_PEERS, channel_id, num_channels, nvl_rank).advance_also(local_buffer_ptr);

        // Get tasks for each RDMA lane
        int token_start_idx = 0, token_end_idx = 0;
        if (lane_id < kNumRDMARanks) {
            int prefix_idx = (lane_id * NUM_MAX_NVL_PEERS + dst_nvl_rank) * num_channels + channel_id;
            token_start_idx = gbl_channel_prefix_matrix[prefix_idx];
            token_end_idx = (prefix_idx == num_channels * num_ranks - 1) ? num_tokens : gbl_channel_prefix_matrix[prefix_idx + 1];
        }
        __syncwarp();

        // NOTES: here the cached value of each lane is only responsible for a single RDMA buffer
        int cached_channel_head_idx = 0, cached_channel_tail_idx = 0;
        EP_STATIC_ASSERT(kNumRDMARanks <= 32, "Invalid number of RDMA peers");

        // Iterate over all tokens and send by chunks
        while (true) {
            // Exit if possible
            if (__all_sync(0xffffffff, token_start_idx >= token_end_idx))
                break;

            // Decide the next RDMA buffer to send
            bool is_lane_ready = false;
            auto start_time = clock64();
            while (true) {
                int num_used_slots = cached_channel_tail_idx - cached_channel_head_idx;
                is_lane_ready = lane_id < kNumRDMARanks and token_start_idx < token_end_idx and num_max_nvl_chunked_recv_tokens_per_rdma - num_used_slots >= num_max_nvl_chunked_send_tokens;
                if (__any_sync(0xffffffff, is_lane_ready))
                    break;

                // Retry
                if (lane_id < kNumRDMARanks and token_start_idx < token_end_idx)
                    cached_channel_head_idx = ld_volatile_global(nvl_channel_head.buffer() + lane_id);

                // Timeout check
                if (clock64() - start_time > NUM_TIMEOUT_CYCLES and lane_id < kNumRDMARanks) {
                    printf("DeepEP combine NVL sender timeout, channel: %d, RDMA: %d, nvl: %d, dst NVL: %d, RDMA lane: %d, head: %d, tail: %d, start: %d, end: %d\n",
                           channel_id, rdma_rank, nvl_rank, dst_nvl_rank, lane_id, ld_volatile_global(nvl_channel_head.buffer() + lane_id), cached_channel_tail_idx,
                           token_start_idx, token_end_idx);
                    trap();
                }
            }

            // Sync token start index and count
            for (int current_rdma_idx = 0; current_rdma_idx < kNumRDMARanks; ++ current_rdma_idx) {
                if (__shfl_sync(0xffffffff, (token_start_idx >= token_end_idx) or (not is_lane_ready), current_rdma_idx))
                    continue;

                // Sync token start index
                auto token_idx = static_cast<int64_t>(__shfl_sync(0xffffffff, token_start_idx, current_rdma_idx));
                int num_tokens_in_chunk = __shfl_sync(0xffffffff, min(num_max_nvl_chunked_send_tokens, token_end_idx - token_start_idx), current_rdma_idx);

                // Send by chunk
                for (int chunk_idx = 0; chunk_idx < num_tokens_in_chunk; ++ chunk_idx, ++ token_idx) {
                    // Get an empty slot
                    int dst_slot_idx = 0;
                    if (lane_id == current_rdma_idx) {
                        dst_slot_idx = (cached_channel_tail_idx ++) % num_max_nvl_chunked_recv_tokens_per_rdma;
                        dst_slot_idx = current_rdma_idx * num_max_nvl_chunked_recv_tokens_per_rdma + dst_slot_idx;
                    }
                    dst_slot_idx = __shfl_sync(0xffffffff, dst_slot_idx, current_rdma_idx);

                    // Copy data
                    auto shifted_x_buffers = nvl_channel_x.buffer() + dst_slot_idx * hidden_int4;
                    auto shifted_x = x + token_idx * hidden_int4;
                    UNROLLED_WARP_COPY(5, lane_id, hidden_int4, shifted_x_buffers, shifted_x, ld_nc_global, st_na_global);

                    // Copy source meta
                    if (lane_id == 0)
                        st_na_global(nvl_channel_src_meta.buffer() + dst_slot_idx, ld_nc_global(src_meta + token_idx));

                    // Copy `topk_weights`
                    if (lane_id < num_topk)
                        st_na_global(nvl_channel_topk_weights.buffer() + dst_slot_idx * num_topk + lane_id, ld_nc_global(topk_weights + token_idx * num_topk + lane_id));
                }
                lane_id == current_rdma_idx ? (token_start_idx = static_cast<int>(token_idx)) : 0;
            }

            // Move queue tail
            __syncwarp();
            if (lane_id < kNumRDMARanks and is_lane_ready)
                st_release_sys_global(nvl_channel_tail.buffer() + lane_id, cached_channel_tail_idx);
        }
    } else {
        // Combiners and coordinators
        // RDMA symmetric layout
        auto hidden_bytes = hidden_int4 * sizeof(int4);
        auto num_bytes_per_rdma_token = get_num_bytes_per_rdma_token(hidden_int4, 0, 0, num_topk);
        auto rdma_channel_data = SymBuffer<int8_t>(rdma_buffer_ptr, num_max_rdma_chunked_recv_tokens * num_bytes_per_rdma_token, kNumRDMARanks, channel_id, num_channels);
        auto rdma_channel_head = SymBuffer<uint64_t, false>(rdma_buffer_ptr, 1, kNumRDMARanks, channel_id, num_channels);
        auto rdma_channel_tail = SymBuffer<uint64_t, false>(rdma_buffer_ptr, 1, kNumRDMARanks, channel_id, num_channels);

        // NVL layouts
        void* local_nvl_buffer = buffer_ptrs[nvl_rank];
        void* nvl_buffers[NUM_MAX_NVL_PEERS];
        #pragma unroll
        for (int i = 0; i < NUM_MAX_NVL_PEERS; ++ i)
            nvl_buffers[i] = buffer_ptrs[i];
        auto nvl_channel_x = AsymBuffer<int4>(local_nvl_buffer, num_max_nvl_chunked_recv_tokens * hidden_int4, NUM_MAX_NVL_PEERS, channel_id, num_channels).advance_also<NUM_MAX_NVL_PEERS>(nvl_buffers);
        auto nvl_channel_src_meta = AsymBuffer<SourceMeta>(local_nvl_buffer, num_max_nvl_chunked_recv_tokens, NUM_MAX_NVL_PEERS, channel_id, num_channels).advance_also<NUM_MAX_NVL_PEERS>(nvl_buffers);
        auto nvl_channel_topk_weights = AsymBuffer<float>(local_nvl_buffer, num_max_nvl_chunked_recv_tokens * num_topk, NUM_MAX_NVL_PEERS, channel_id, num_channels).advance_also<NUM_MAX_NVL_PEERS>(nvl_buffers);
        auto nvl_channel_head = AsymBuffer<int, NUM_MAX_NVL_PEERS>(nvl_buffers, kNumRDMARanks, NUM_MAX_NVL_PEERS, channel_id, num_channels, nvl_rank).advance_also(local_nvl_buffer);
        auto nvl_channel_tail = AsymBuffer<int>(local_nvl_buffer, kNumRDMARanks, NUM_MAX_NVL_PEERS, channel_id, num_channels).advance_also<NUM_MAX_NVL_PEERS>(nvl_buffers);

        // Combiner warp synchronization
        __shared__ volatile int forwarder_nvl_head[kNumForwarders][NUM_MAX_NVL_PEERS];
        __shared__ volatile bool forwarder_retired[kNumForwarders];
        __shared__ volatile int rdma_receiver_rdma_head[kNumRDMAReceivers][kNumRDMARanks];
        __shared__ volatile bool rdma_receiver_retired[kNumRDMAReceivers];
        auto sync_forwarder_smem = [=]() { asm volatile("bar.sync 0, %0;" :: "r"((kNumForwarders + 1) * 32)); };
        auto sync_rdma_receiver_smem = [=]() { asm volatile("bar.sync 1, %0;" :: "r"((kNumRDMAReceivers + 1) * 32)); };

        if (warp_role == WarpRole::kNVLAndRDMAForwarder) {
            // Receive from NVL ranks and forward to RDMA ranks
            // NOTES: this part is using "large warps" for each RDMA ranks
            const auto dst_rdma_rank = warp_id / kNumWarpsPerForwarder;
            const auto sub_warp_id = warp_id % kNumWarpsPerForwarder;
            auto send_buffer = dst_rdma_rank == rdma_rank ? rdma_channel_data.recv_buffer(dst_rdma_rank) : rdma_channel_data.send_buffer(dst_rdma_rank);
            auto sync_large_warp = [=]() {
                if (kNumWarpsPerForwarder == 1) {
                    __syncwarp();
                } else {
                    asm volatile("bar.sync %0, %1;" :: "r"(dst_rdma_rank + 2), "r"(kNumWarpsPerForwarder * 32));
                }
            };
            EP_STATIC_ASSERT(kNumWarpsPerForwarder == 1 or kNumRDMARanks + 2 <= 16, "Barriers are not enough");

            // Advance to the corresponding NVL buffer
            nvl_channel_x.advance(dst_rdma_rank * num_max_nvl_chunked_recv_tokens_per_rdma * hidden_int4);
            nvl_channel_src_meta.advance(dst_rdma_rank * num_max_nvl_chunked_recv_tokens_per_rdma);
            nvl_channel_topk_weights.advance(dst_rdma_rank * num_max_nvl_chunked_recv_tokens_per_rdma * num_topk);
            nvl_channel_head.advance(dst_rdma_rank);
            nvl_channel_tail.advance(dst_rdma_rank);

            // Clean shared memory and sync
            EP_STATIC_ASSERT(NUM_MAX_NVL_PEERS <= 32, "Invalid number of NVL peers");
            lane_id < NUM_MAX_NVL_PEERS ? (forwarder_nvl_head[warp_id][lane_id] = 0) : 0;
            lane_id == 0 ? (forwarder_retired[warp_id] = false) : false;
            sync_forwarder_smem();

            // Get count and cached head
            int cached_nvl_channel_tail_idx = 0;
            int num_tokens_to_combine = rdma_channel_prefix_matrix[dst_rdma_rank * num_channels + channel_id];
            int num_tokens_prefix = channel_id == 0 ? 0 : rdma_channel_prefix_matrix[dst_rdma_rank * num_channels + channel_id - 1];
            num_tokens_to_combine -= num_tokens_prefix;
            num_tokens_prefix += dst_rdma_rank == 0 ? 0 : rdma_rank_prefix_sum[dst_rdma_rank - 1];
            combined_nvl_head += num_tokens_prefix * NUM_MAX_NVL_PEERS;

            // Iterate over all tokens and combine by chunks
            for (int token_start_idx = 0; token_start_idx < num_tokens_to_combine; token_start_idx += num_max_rdma_chunked_send_tokens) {
                // Check destination queue emptiness, or wait a buffer to be released
                auto token_end_idx = min(token_start_idx + num_max_rdma_chunked_send_tokens, num_tokens_to_combine);
                auto num_chunked_tokens = token_end_idx - token_start_idx;
                auto start_time = clock64();
                while (sub_warp_id == 0 and lane_id == 0) {
                    // Inequality: `num_max_rdma_chunked_recv_tokens - (tail - head) >= num_chunked_tokens`
                    // Here, `token_start_idx` is the actual tail
                    int num_used_slots = token_start_idx - ld_volatile_global(rdma_channel_head.buffer(dst_rdma_rank));
                    if (num_max_rdma_chunked_recv_tokens - num_used_slots >= num_chunked_tokens)
                        break;

                    // Timeout check
                    if (clock64() - start_time > NUM_TIMEOUT_CYCLES) {
                        printf("DeepEP combine forwarder (RDMA check) timeout, channel: %d, RDMA: %d, nvl: %d, dst RDMA: %d, head: %ld, tail: %d, chunked: %d\n",
                               channel_id, rdma_rank, nvl_rank, dst_rdma_rank, ld_volatile_global(rdma_channel_head.buffer(dst_rdma_rank)), token_start_idx, num_chunked_tokens);
                        trap();
                    }
                }
                sync_large_warp();

                // Combine and write to the RDMA buffer
                for (int token_idx = token_start_idx + sub_warp_id; token_idx < token_end_idx; token_idx += kNumWarpsPerForwarder) {
                    // Read expected head
                    EP_STATIC_ASSERT(kNumRDMARanks <= 32, "Invalid number of RDMA peers");
                    int expected_head = -1;
                    if (lane_id < NUM_MAX_NVL_PEERS)
                        expected_head = ld_nc_global(combined_nvl_head + token_idx * NUM_MAX_NVL_PEERS + lane_id);

                    // Wait lanes to be ready
                    start_time = clock64();
                    while (cached_nvl_channel_tail_idx <= expected_head) {
                        cached_nvl_channel_tail_idx = ld_acquire_sys_global(nvl_channel_tail.buffer(lane_id));

                        // Timeout check
                        if (clock64() - start_time > NUM_TIMEOUT_CYCLES and lane_id < NUM_MAX_NVL_PEERS) {
                            printf("DeepEP combine forwarder (NVL check) timeout, channel: %d, RDMA: %d, nvl: %d, src NVL: %d, dst RDMA: %d, tail: %d, waiting: %d, total: %d, sub: %d, large: %d, expected: %d\n",
                                   channel_id, rdma_rank, nvl_rank, lane_id, dst_rdma_rank, cached_nvl_channel_tail_idx, token_idx, num_tokens_to_combine, sub_warp_id, kNumWarpsPerForwarder, expected_head);
                            trap();
                        }
                    }

                    // Combine current token
                    auto rdma_slot_idx = token_idx % num_max_rdma_chunked_recv_tokens;
                    void* shifted = send_buffer + rdma_slot_idx * num_bytes_per_rdma_token;
                    auto recv_fn = [&](int src_nvl_rank, int slot_idx, int hidden_int4_idx) -> int4 { return ld_nc_global(nvl_channel_x.buffer(src_nvl_rank) + slot_idx * hidden_int4 + hidden_int4_idx); };
                    auto recv_tw_fn = [&](int src_nvl_rank, int slot_idx, int topk_idx) -> float { return ld_nc_global(nvl_channel_topk_weights.buffer(src_nvl_rank) + slot_idx * num_topk + topk_idx); };
                    combine_token<NUM_MAX_NVL_PEERS, false, dtype_t, NUM_MAX_NVL_PEERS>(expected_head >= 0,
                                                                                 expected_head, lane_id,
                                                                                 hidden_int4, num_topk,
                                                                                 static_cast<int4*>(shifted),
                                                                                 reinterpret_cast<float*>(static_cast<int8_t*>(shifted) + hidden_bytes + sizeof(SourceMeta)),
                                                                                 nullptr, nullptr, num_max_nvl_chunked_recv_tokens_per_rdma, recv_fn, recv_tw_fn);

                    // Update head
                    if (lane_id < NUM_MAX_NVL_PEERS)
                        expected_head < 0 ? (forwarder_nvl_head[warp_id][lane_id] = -expected_head - 1) : (forwarder_nvl_head[warp_id][lane_id] = expected_head + 1);
                }
                sync_large_warp();

                // Issue RDMA send
                if (sub_warp_id == kNumWarpsPerForwarder - 1) {
                    if (dst_rdma_rank != rdma_rank) {
                        auto rdma_slot_idx = token_start_idx % num_max_rdma_chunked_recv_tokens;
                        const size_t num_bytes_per_msg = num_chunked_tokens * num_bytes_per_rdma_token;
                        const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_channel_data.recv_buffer(rdma_rank) + rdma_slot_idx * num_bytes_per_rdma_token);
                        const auto src_ptr = reinterpret_cast<uint64_t>(rdma_channel_data.send_buffer(dst_rdma_rank) + rdma_slot_idx * num_bytes_per_rdma_token);
                        nvshmemi_ibgda_put_nbi_warp<true>(dst_ptr, src_ptr, num_bytes_per_msg,
                                                          translate_dst_rdma_rank<kLowLatencyMode>(dst_rdma_rank, nvl_rank), channel_id, lane_id, 0);
                    } else {
                        memory_fence();
                    }

                    // Write new RDMA tail
                    __syncwarp();
                    if (lane_id == 0) {
                        nvshmemi_ibgda_amo_nonfetch_add(rdma_channel_tail.buffer(rdma_rank), num_chunked_tokens,
                                                        translate_dst_rdma_rank<kLowLatencyMode>(dst_rdma_rank, nvl_rank), channel_id, dst_rdma_rank == rdma_rank);
                    }
                }
            }

            // Retired
            __syncwarp();
            if (lane_id == 0)
                forwarder_retired[warp_id] = true;
        } else if (warp_role == WarpRole::kRDMAReceiver) {
            // Receive from RDMA ranks and write to the output tensor
            // Clean shared memory and sync
            EP_DEVICE_ASSERT(kNumRDMARanks <= 32);
            lane_id < kNumRDMARanks ? (rdma_receiver_rdma_head[warp_id][lane_id] = 0) : 0;
            lane_id == 0 ? (rdma_receiver_retired[warp_id] = false) : 0;
            sync_rdma_receiver_smem();

            // The same tokens as the dispatch process
            int token_start_idx, token_end_idx;
            get_channel_task_range(num_combined_tokens, num_channels, channel_id, token_start_idx, token_end_idx);

            // Iterate over all tokens and combine
            int cached_channel_tail_idx = 0;
            for (int64_t token_idx = token_start_idx + warp_id; token_idx < token_end_idx; token_idx += kNumRDMAReceivers) {
                // Read expected head
                EP_STATIC_ASSERT(kNumRDMARanks <= 32, "Invalid number of RDMA peers");
                int expected_head = -1;
                if (lane_id < kNumRDMARanks) {
                    expected_head = ld_nc_global(combined_rdma_head + token_idx * kNumRDMARanks + lane_id);
                    (expected_head < 0) ? (rdma_receiver_rdma_head[warp_id][lane_id] = -expected_head - 1) : (rdma_receiver_rdma_head[warp_id][lane_id] = expected_head);
                }

                // Wait lanes to be ready
                auto start_time = clock64();
                while (cached_channel_tail_idx <= expected_head) {
                    cached_channel_tail_idx = static_cast<int>(ld_acquire_sys_global(rdma_channel_tail.buffer(lane_id)));

                    // Timeout check
                    if (clock64() - start_time > NUM_TIMEOUT_CYCLES) {
                        printf("DeepEP combine RDMA receiver timeout, channel: %d, RDMA: %d, nvl: %d, src RDMA: %d, tail: %d, waiting: %ld, expect: %d\n",
                               channel_id, rdma_rank, nvl_rank, lane_id, cached_channel_tail_idx, token_idx, expected_head);
                        trap();
                    }
                }
                __syncwarp();

                // Combine current token
                auto recv_fn = [&](int src_rdma_rank, int slot_idx, int hidden_int4_idx) -> int4 { return ld_nc_global(reinterpret_cast<const int4*>(rdma_channel_data.recv_buffer(src_rdma_rank) + slot_idx * num_bytes_per_rdma_token) + hidden_int4_idx);};
                auto recv_tw_fn = [&](int src_rdma_rank, int slot_idx, int topk_idx) -> float { return ld_nc_global(reinterpret_cast<const float*>(rdma_channel_data.recv_buffer(src_rdma_rank) + slot_idx * num_bytes_per_rdma_token + hidden_bytes + sizeof(SourceMeta)) + topk_idx);};
                combine_token<kNumRDMARanks, true, dtype_t, kNumTopkRDMARanks>(expected_head >= 0,
                                                                         expected_head, lane_id,
                                                                         hidden_int4, num_topk,
                                                                         combined_x + token_idx * hidden_int4,
                                                                         combined_topk_weights + token_idx * num_topk,
                                                                         bias_0 == nullptr ? nullptr : bias_0 + token_idx * hidden_int4,
                                                                         bias_1 == nullptr ? nullptr : bias_1 + token_idx * hidden_int4,
                                                                         num_max_rdma_chunked_recv_tokens, recv_fn, recv_tw_fn);
            }

            // Retired
            __syncwarp();
            if (lane_id == 0)
                rdma_receiver_retired[warp_id] = true;
        } else {
            // Coordinator
            // Sync shared memory status
            is_rdma_receiver_sm ? sync_rdma_receiver_smem() : sync_forwarder_smem();
            const auto num_warps_per_rdma_rank = kNumForwarders / kNumRDMARanks;

            int last_rdma_head = 0;
            int last_nvl_head[kNumRDMARanks] = {0};
            int dst_rdma_rank = lane_id < kNumRDMARanks ? lane_id : 0;
            int dst_nvl_rank = lane_id < NUM_MAX_NVL_PEERS ? lane_id : 0;
            EP_STATIC_ASSERT(kNumCombineForwarderWarps <= 32, "Invalid number of forwarder warps");
            while (true) {
                // Retired
                if (is_rdma_receiver_sm and __all_sync(0xffffffff, lane_id >= kNumRDMAReceivers or rdma_receiver_retired[lane_id]))
                    break;
                if (not is_rdma_receiver_sm and __all_sync(0xffffffff, lane_id >= kNumForwarders or forwarder_retired[lane_id]))
                    break;

                // Find minimum head for RDMA ranks
                if (is_rdma_receiver_sm) {
                    int min_head = std::numeric_limits<int>::max();
                    #pragma unroll
                    for (int i = 0; i < kNumRDMAReceivers; ++ i) if (not rdma_receiver_retired[i])
                        min_head = min(min_head, rdma_receiver_rdma_head[i][dst_rdma_rank]);
                    if (min_head != std::numeric_limits<int>::max() and min_head >= last_rdma_head + num_max_rdma_chunked_send_tokens and lane_id < kNumRDMARanks) {
                        nvshmemi_ibgda_amo_nonfetch_add(rdma_channel_head.buffer(rdma_rank), min_head - last_rdma_head,
                                                        translate_dst_rdma_rank<kLowLatencyMode>(dst_rdma_rank, nvl_rank), channel_id + num_channels, dst_rdma_rank == rdma_rank);
                        last_rdma_head = min_head;
                    }
                } else {
                    // Find minimum head for NVL ranks
                    #pragma unroll
                    for (int i = 0; i < kNumRDMARanks; ++ i) {
                        int min_head = std::numeric_limits<int>::max();
                        #pragma unroll
                        for (int j = 0; j < num_warps_per_rdma_rank; ++ j) if (not forwarder_retired[i * num_warps_per_rdma_rank + j])
                            min_head = min(min_head, forwarder_nvl_head[i * num_warps_per_rdma_rank + j][dst_nvl_rank]);
                        if (min_head != std::numeric_limits<int>::max() and min_head > last_nvl_head[i] and lane_id < NUM_MAX_NVL_PEERS)
                            st_relaxed_sys_global(nvl_channel_head.buffer_by(dst_nvl_rank) + i, last_nvl_head[i] = min_head);
                    }
                }

                // Nanosleep and let other warps work
                __nanosleep(NUM_WAIT_NANOSECONDS);
            }
        }
    }
}

void combine(hipDataType type,
             void* combined_x, float* combined_topk_weights,
             const bool* is_combined_token_in_rank,
             const void* x, const float* topk_weights,
             const void* bias_0, const void* bias_1,
             const int* combined_rdma_head, const int* combined_nvl_head,
             const void* src_meta, const int* rdma_channel_prefix_matrix, const int* rdma_rank_prefix_sum, const int* gbl_channel_prefix_matrix,
             int num_tokens, int num_combined_tokens, int hidden, int num_topk,
             void* rdma_buffer_ptr, int num_max_rdma_chunked_send_tokens, int num_max_rdma_chunked_recv_tokens,
             void** buffer_ptrs, int num_max_nvl_chunked_send_tokens, int num_max_nvl_chunked_recv_tokens,
             int rank, int num_ranks, hipStream_t stream, int num_channels, bool low_latency_mode) {
    constexpr int kNumCombineForwarderWarps = 16;

#define COMBINE_LAUNCH_CASE(num_rdma_ranks) { \
    auto combine_func = low_latency_mode ? \
        combine<true, num_rdma_ranks, hip_bfloat16, kNumCombineForwarderWarps> : combine<false, num_rdma_ranks, hip_bfloat16, kNumCombineForwarderWarps>; \
    LAUNCH_KERNEL(&cfg, combine_func, \
                  reinterpret_cast<int4*>(combined_x), combined_topk_weights, is_combined_token_in_rank, \
                  reinterpret_cast<const int4*>(x), topk_weights, \
                  reinterpret_cast<const int4*>(bias_0), reinterpret_cast<const int4*>(bias_1), \
                  combined_rdma_head, combined_nvl_head, \
                  reinterpret_cast<const SourceMeta*>(src_meta), rdma_channel_prefix_matrix, rdma_rank_prefix_sum, gbl_channel_prefix_matrix, \
                  num_tokens, num_combined_tokens, hidden, num_topk, \
                  rdma_buffer_ptr, num_max_rdma_chunked_send_tokens, num_max_rdma_chunked_recv_tokens, \
                  buffer_ptrs, num_max_nvl_chunked_send_tokens, num_max_nvl_chunked_recv_tokens, \
                  rank, num_ranks); } break

    int num_rdma_ranks = num_ranks / NUM_MAX_NVL_PEERS;
    auto num_warps_per_forwarder = std::max(kNumCombineForwarderWarps / num_rdma_ranks, 1);
    int num_forwarder_warps = num_rdma_ranks * num_warps_per_forwarder;
    EP_HOST_ASSERT(num_forwarder_warps > 0 and num_forwarder_warps % num_rdma_ranks == 0);
    EP_HOST_ASSERT(num_max_nvl_chunked_recv_tokens % num_rdma_ranks == 0);
    EP_HOST_ASSERT(num_max_nvl_chunked_recv_tokens / num_rdma_ranks > std::max(num_max_rdma_chunked_send_tokens, num_max_nvl_chunked_send_tokens));
    EP_HOST_ASSERT(type == HIP_R_16BF);

    SETUP_LAUNCH_CONFIG(num_channels * 2, (NUM_MAX_NVL_PEERS + num_forwarder_warps + 1) * 32, stream);
    SWITCH_RDMA_RANKS(COMBINE_LAUNCH_CASE);
#undef COMBINE_LAUNCH_CASE
}

} // namespace internode

} // namespace deep_ep
