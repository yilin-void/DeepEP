#include "hip/hip_runtime.h"
#include "configs.cuh"
#include "exception.cuh"
#include "launch.cuh"

namespace deep_ep {

namespace layout {

template <int kNumThreads, int kNumExpertsPerSM, int kNumRanksPerSM>
__global__ void __launch_bounds__(kNumThreads, 1)
get_dispatch_layout(const int* topk_idx,
                    int* num_tokens_per_rank, int* num_tokens_per_rdma_rank,
                    int* num_tokens_per_expert, bool* is_token_in_rank,
                    int num_tokens, int num_topk, int num_ranks, int num_experts) {
    auto sm_id = static_cast<int>(blockIdx.x);
    auto thread_id = static_cast<int>(threadIdx.x);

    // Count expert statistics
    __shared__ int num_tokens_per_expert_per_thread[kNumThreads][kNumExpertsPerSM];
    int expert_begin_idx = sm_id * kNumExpertsPerSM, expert_end_idx = min(expert_begin_idx + kNumExpertsPerSM, num_experts);
    if (expert_begin_idx < expert_end_idx) {
        // Per-thread count
        #pragma unroll
        for (int i = 0; i < kNumExpertsPerSM; ++ i)
            num_tokens_per_expert_per_thread[thread_id][i] = 0;
        #pragma unroll
        for (int i = thread_id; i < num_tokens; i += kNumThreads) {
            auto shifted_topk_idx = topk_idx + i * num_topk;
            #pragma unroll
            for (int j = 0, expert_idx; j < num_topk; ++ j) {
                expert_idx = static_cast<int>(shifted_topk_idx[j]);
                if (expert_begin_idx <= expert_idx and expert_idx < expert_end_idx)
                    ++ num_tokens_per_expert_per_thread[thread_id][expert_idx - expert_begin_idx];
            }
        }
        __syncthreads();

        // Sum up
        EP_STATIC_ASSERT(kNumExpertsPerSM <= kNumThreads, "Too many experts per SM");
        if (expert_begin_idx + thread_id < expert_end_idx) {
            int sum = 0;
            #pragma unroll
            for (int i = 0; i < kNumThreads; ++ i)
                sum += num_tokens_per_expert_per_thread[i][thread_id];
            num_tokens_per_expert[expert_begin_idx + thread_id] = sum;
        }
        return;
    }

    if (num_tokens_per_rdma_rank != nullptr)
        EP_DEVICE_ASSERT(num_ranks % NUM_MAX_NVL_PEERS == 0 and num_ranks > NUM_MAX_NVL_PEERS);

    // Count rank statistics
    constexpr int kNumRDMARanksPerSM = kNumRanksPerSM / NUM_MAX_NVL_PEERS;
    __shared__ int num_tokens_per_rank_per_thread[kNumThreads][kNumRanksPerSM];
    __shared__ int num_tokens_per_rdma_rank_per_thread[kNumThreads][kNumRDMARanksPerSM];
    auto sm_begin = (num_experts + kNumExpertsPerSM - 1) / kNumExpertsPerSM;
    int rank_begin_idx = (sm_id - sm_begin) * kNumRanksPerSM, rank_end_idx = min(rank_begin_idx + kNumRanksPerSM, num_ranks);
    int rdma_rank_begin_idx = rank_begin_idx / NUM_MAX_NVL_PEERS, rdma_rank_end_idx = rank_end_idx / NUM_MAX_NVL_PEERS;
    if (rank_begin_idx < rank_end_idx) {
        const auto num_expert_per_rank = num_experts / num_ranks;
        auto expert_begin = rank_begin_idx * num_expert_per_rank;
        auto expert_end = rank_end_idx * num_expert_per_rank;

        // Per-thread count
        #pragma unroll
        for (int i = 0; i < kNumRanksPerSM; ++ i)
            num_tokens_per_rank_per_thread[thread_id][i] = 0;
        #pragma unroll
        for (int i = 0; i < kNumRDMARanksPerSM; ++ i)
            num_tokens_per_rdma_rank_per_thread[thread_id][i] = 0;
        #pragma unroll
        for (int i = thread_id; i < num_tokens; i += kNumThreads) {
            auto shifted_topk_idx = topk_idx + i * num_topk;
            int is_in_rank[kNumRanksPerSM] = {0}, is_in_rdma_rank[kNumRDMARanksPerSM] = {0};
            #pragma unroll
            for (int j = 0, expert_idx, rank_idx; j < num_topk; ++j) {
                expert_idx = static_cast<int>(shifted_topk_idx[j]);
                if (expert_begin <= expert_idx and expert_idx < expert_end) {
                    // Count single rank
                    rank_idx = expert_idx / num_expert_per_rank - rank_begin_idx;
                    is_in_rank[rank_idx] ++, is_in_rdma_rank[rank_idx / NUM_MAX_NVL_PEERS] ++;
                }
            }

            auto shifted_is_token_in_rank = is_token_in_rank + i * num_ranks;
            #pragma unroll
            for (int j = 0; j + rank_begin_idx < rank_end_idx; ++ j) {
                shifted_is_token_in_rank[j + rank_begin_idx] = (is_in_rank[j] > 0);
                num_tokens_per_rank_per_thread[thread_id][j] += (is_in_rank[j] > 0);
            }

            #pragma unroll
            for (int j = 0; j + rdma_rank_begin_idx < rdma_rank_end_idx; ++ j)
                num_tokens_per_rdma_rank_per_thread[thread_id][j] += (is_in_rdma_rank[j] > 0);
        }
        __syncthreads();

        // Sum up
        EP_STATIC_ASSERT(kNumRanksPerSM <= kNumThreads, "Too many ranks per SM");
        if (rank_begin_idx + thread_id < rank_end_idx) {
            int sum = 0;
            #pragma unroll
            for (int i = 0; i < kNumThreads; ++ i)
                sum += num_tokens_per_rank_per_thread[i][thread_id];
            num_tokens_per_rank[rank_begin_idx + thread_id] = sum;
        }

        if (num_tokens_per_rdma_rank != nullptr and rdma_rank_begin_idx + thread_id < rdma_rank_end_idx) {
            int sum = 0;
            #pragma unroll
            for (int i = 0; i < kNumThreads; ++ i)
                sum += num_tokens_per_rdma_rank_per_thread[i][thread_id];
            num_tokens_per_rdma_rank[rdma_rank_begin_idx + thread_id] = sum;
        }
    }
}

void get_dispatch_layout(const int* topk_idx,
                         int* num_tokens_per_rank, int* num_tokens_per_rdma_rank,
                         int* num_tokens_per_expert, bool* is_token_in_rank,
                         int num_tokens, int num_topk, int num_ranks, int num_experts,
                         hipStream_t stream) {
    constexpr int kNumThreads = 256, kNumExpertsPerSM = 32, kNumRanksPerSM = 8;
    int num_sms = ((num_experts + kNumExpertsPerSM - 1) / kNumExpertsPerSM) + (num_ranks + kNumRanksPerSM - 1) / kNumRanksPerSM;
    EP_STATIC_ASSERT(kNumExpertsPerSM % NUM_MAX_NVL_PEERS == 0, "Invalid number of experts per SM");

    SETUP_LAUNCH_CONFIG(num_sms, kNumThreads, stream);
    LAUNCH_KERNEL(&cfg, (get_dispatch_layout<kNumThreads, kNumExpertsPerSM, kNumRanksPerSM>),
                  topk_idx, num_tokens_per_rank, num_tokens_per_rdma_rank, num_tokens_per_expert, is_token_in_rank,
                  num_tokens, num_topk, num_ranks, num_experts);
}

} // namespace layout

} // namespace deep_ep
