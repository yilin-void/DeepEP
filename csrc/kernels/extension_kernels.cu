#include "hip/hip_runtime.h"
#include "configs.cuh"
#include "exception.cuh"
#include "launch.cuh"
#include "ibgda_device.cuh"

namespace deep_ep {

namespace extensions {

template <int kHidden>
__global__ __launch_bounds__(1024, 1) void
dispatch_fp4(void* packed_recv_x, void* packed_recv_x_scales,
            int* packed_recv_src_info, int64_t* packed_recv_layout_range,
            int* packed_recv_count,
            int* cumulative_local_expert_recv_stats,
            void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
            const void* x, const void* x_scales, const int* topk_idx,
            int* atomic_counter_per_expert, int* atomic_finish_counter_per_expert,
            int* next_clean, int num_next_clean_int,
            int num_tokens, int num_max_dispatch_tokens_per_rank,
            int num_topk, int num_experts, int rank, int num_ranks,
            int num_warp_groups, int num_warps_per_group, int phases) {
    EP_STATIC_ASSERT(kHidden % 32 == 0, "Invalid hidden size");
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto warp_id = thread_id / 32, lane_id = get_lane_id();
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

    const size_t hidden_bytes = kHidden / 2;
    const size_t hidden_int4 = hidden_bytes / sizeof(int4);
    const size_t scales_bytes = kHidden / 16;
    const size_t scales_int4 = scales_bytes / sizeof(int4);

    using vec_t = int4;
    const size_t num_bytes_per_msg = sizeof(int4) + hidden_bytes + kHidden / 16;
    const size_t num_int4_per_msg = num_bytes_per_msg / sizeof(int4);
    EP_DEVICE_ASSERT(num_bytes_per_msg % sizeof(int4) == 0);

    // Expert counts
    constexpr int kNumMaxWarpGroups = 32;
    __shared__ int shared_num_tokens_sent_per_expert[kNumMaxWarpGroups];

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_DISPATCH_RECV;

    // There are 2 kinds of warps in this part:
    // 1. The first-kind warps for sending top-k tokens
    // 2. The last warp for reading `topk_idx` and count for per-expert information
    if (warp_id < num_warps - 1) {
        const auto num_threads = (num_warps - 1) * 32;

        for (int token_idx = sm_id; token_idx < num_tokens; token_idx += num_sms) {
            const auto x_int4 = static_cast<const int4*>(x) + token_idx * hidden_int4;
            const auto x_scales_int4 = static_cast<const int4*>(x_scales) + token_idx * scales_int4;
            const auto rdma_x_src_idx = reinterpret_cast<int*>(static_cast<uint8_t*>(rdma_x) + token_idx * num_bytes_per_msg);
            const auto rdma_x_vec = reinterpret_cast<vec_t*>(reinterpret_cast<uint8_t*>(rdma_x_src_idx) + sizeof(int4));
            const auto rdma_x_scales = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(rdma_x_vec) + hidden_bytes);

            // Overlap top-k index read and source token index writes
            auto dst_expert_idx = warp_id < num_topk ? static_cast<int>(__ldg(topk_idx + token_idx * num_topk + warp_id)) : -1;
            thread_id == 0 ? (*rdma_x_src_idx = token_idx) : 0;

            #pragma unroll
            for (int i = thread_id; i < hidden_int4; i += num_threads) {
                auto int4_value = __ldg(x_int4 + i);
                rdma_x_vec[i] = *reinterpret_cast<vec_t*>(&int4_value);
            }
            #pragma unroll
            for (int i = thread_id; i < scales_int4; i += num_threads) {
                auto int4_value = __ldg(x_scales_int4 + i);
                rdma_x_scales[i] = *reinterpret_cast<int4*>(&int4_value);
            }
            asm volatile("bar.sync 1, %0;" :: "r"(num_threads));

            // Issue IBGDA sends
            if (dst_expert_idx >= 0) {
                int slot_idx = lane_id == 0 ? atomicAdd(atomic_counter_per_expert + dst_expert_idx, 1) : 0;
                slot_idx = __shfl_sync(0xffffffff, slot_idx, 0);
                const auto dst_rank = dst_expert_idx / num_local_experts;
                const auto dst_expert_local_idx = dst_expert_idx % num_local_experts;
                const auto src_ptr = reinterpret_cast<uint64_t>(rdma_x_src_idx);
                const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) +
                                        dst_expert_local_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                                        rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                                        slot_idx * num_bytes_per_msg;
                const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
                if (dst_p2p_ptr == 0) {
                    nvshmemi_ibgda_put_nbi_warp(dst_ptr, src_ptr, num_bytes_per_msg, dst_rank, dst_expert_local_idx, lane_id, slot_idx);
                } else {
                    constexpr int kUnrollFactor = ((kHidden / 2 + kHidden / 16) / sizeof(int4) + 31) / 32;
                    EP_STATIC_ASSERT(kUnrollFactor > 0, "Invalid unroll factor");
                    const auto* src_int4_ptr = reinterpret_cast<const int4*>(src_ptr);
                    const auto* dst_int4_ptr = reinterpret_cast<int4*>(dst_p2p_ptr);
                    UNROLLED_WARP_COPY(kUnrollFactor, lane_id, num_int4_per_msg, dst_int4_ptr, src_int4_ptr, ld_nc_global, st_na_global);
                }

                // Increase counter after finishing
                __syncwarp();
                lane_id == 0 ? atomic_add_release_global(atomic_finish_counter_per_expert + dst_expert_idx, 1) : 0;
            }
        }
    } else if (warp_id == num_warps - 1) {
        EP_DEVICE_ASSERT(num_sms > 1);
        if (sm_id == 0) {
            // The first SM is also responsible for checking QPs
            EP_DEVICE_ASSERT(ibgda_get_state()->num_rc_per_pe >= num_local_experts);

            // The first SM is also responsible for cleaning the next buffer
            #pragma unroll
            for (int i = lane_id; i < num_next_clean_int; i += 32)
                next_clean[i] = 0;

            // Notify before executing `int_p`
            __syncwarp();
            #pragma unroll
            for (int i = lane_id; i < num_experts; i += 32)
                atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG);
        }

        // This SM should be responsible for some destination experts, read `topk_idx` for them
        int expert_count[kNumMaxWarpGroups] = {0};
        const auto expert_begin_idx = sm_id * num_warp_groups;
        const auto expert_end_idx = min(expert_begin_idx + num_warp_groups, num_experts);

        // Per lane count
        #pragma unroll 8
        for (int i = lane_id; i < num_tokens * num_topk; i += 32) {
            auto idx = static_cast<int>(__ldg(topk_idx + i));
            if (idx >= expert_begin_idx and idx < expert_end_idx)
                expert_count[idx - expert_begin_idx] ++;
        }

        // Warp reduce
        #pragma unroll
        for (int i = expert_begin_idx; i < expert_end_idx; ++ i) {
            auto sum = warp_reduce_sum(expert_count[i - expert_begin_idx]);
            if (lane_id == 0) {
                shared_num_tokens_sent_per_expert[i - expert_begin_idx] = sum;
                atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG - sum);
            }
        }
    }
    __syncthreads();

    // Issue count sends
    if (responsible_expert_idx < num_experts and sub_warp_id == 0 and lane_id == 0) {
        const auto dst_rank = responsible_expert_idx / num_local_experts;
        const auto dst_expert_local_idx = responsible_expert_idx % num_local_experts;
        const auto num_tokens_sent = shared_num_tokens_sent_per_expert[responsible_expert_idx - sm_id * num_warp_groups];

        // Wait local sends issued and send expert counts
        while (ld_acquire_global(atomic_finish_counter_per_expert + responsible_expert_idx) != FINISHED_SUM_TAG * 2);
        auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_count + dst_expert_local_idx * num_ranks + rank);
        auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
        if (dst_p2p_ptr == 0) {
            nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(dst_ptr), -num_tokens_sent - 1, dst_rank, dst_expert_local_idx);
        } else {
            st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), -num_tokens_sent - 1);
        }

        // Clean workspace for next use
        atomic_counter_per_expert[responsible_expert_idx] = 0;
        atomic_finish_counter_per_expert[responsible_expert_idx] = 0;

        // Clean `packed_recv_count`
        if (dst_rank == 0)
            packed_recv_count[dst_expert_local_idx] = 0;
    }
    __syncwarp();

    // Receiving phase
    LOW_LATENCY_DISPATCH_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;

    // For send-and-recv kernels, we need a grid sync for making `packed_recv_count` visible
    if (phases & LOW_LATENCY_SEND_PHASE)
        cg::this_grid().sync();

    // Receiving and packing
    if (responsible_expert_idx < num_experts) {
        const auto src_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto rdma_recv_x_uint8 = static_cast<uint8_t*>(rdma_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                src_rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg;
        const auto recv_x_int4 = static_cast<int4*>(packed_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_int4;
        const auto recv_src_info = packed_recv_src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto recv_range = packed_recv_layout_range + local_expert_idx * num_ranks;
        const auto recv_scales_int4 = static_cast<int4*>(packed_recv_x_scales) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * scales_int4;

        // Shared between sub-warps in warp groups
        __shared__ int shared_num_recv_tokens[kNumMaxWarpGroups], shared_recv_token_begin_idx[kNumMaxWarpGroups];

        // Wait tokens to arrive
        // NOTES: using sub-warp 1 to overlap with sub-warp 0
        int num_recv_tokens, recv_token_begin_idx;
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 15);
        if (sub_warp_id == 1 and lane_id == 0) {
            while ((num_recv_tokens = ld_acquire_sys_global(rdma_recv_count + local_expert_idx * num_ranks + src_rank)) == 0);
            num_recv_tokens = -num_recv_tokens - 1;
            recv_token_begin_idx = atomicAdd(packed_recv_count + local_expert_idx, num_recv_tokens);
            shared_num_recv_tokens[warp_group_id] = num_recv_tokens;
            shared_recv_token_begin_idx[warp_group_id] = recv_token_begin_idx;
            recv_range[src_rank] = pack2<int, int64_t>(num_recv_tokens, recv_token_begin_idx);
            if (cumulative_local_expert_recv_stats != nullptr)
                atomicAdd(cumulative_local_expert_recv_stats + local_expert_idx, num_recv_tokens);
        }
        asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 2), "r"(num_warps_per_group * 32));
        num_recv_tokens = shared_num_recv_tokens[warp_group_id];
        recv_token_begin_idx = shared_recv_token_begin_idx[warp_group_id];

        // Copy tokens
        for (int i = sub_warp_id; i < num_recv_tokens; i += num_warps_per_group) {
            // Copy source info
            const auto src_src_idx = reinterpret_cast<int*>(rdma_recv_x_uint8 + i * num_bytes_per_msg);
            if (lane_id == 0)
                recv_src_info[recv_token_begin_idx + i] = ld_nc_global(src_src_idx);
            __syncwarp();

            // Copy data
            // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
            const auto src_data = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));
            const auto dst_data = recv_x_int4 + (recv_token_begin_idx + i) * hidden_int4;
            const auto src_scales = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
            const auto dst_scales = recv_scales_int4 + (recv_token_begin_idx + i) * scales_int4;
            constexpr int kUnrollFactorData = ((kHidden / 2) / sizeof(int4) + 31) / 32;
            constexpr int kUnrollFactorScales = ((kHidden / 16) / sizeof(int4) + 31) / 32;
            EP_STATIC_ASSERT(kUnrollFactorData > 0 and kUnrollFactorScales > 0, "Invalid unroll factor");
            UNROLLED_WARP_COPY(kUnrollFactorData, lane_id, hidden_int4, dst_data, src_data, ld_nc_global, st_na_global);
            UNROLLED_WARP_COPY(kUnrollFactorScales, lane_id, scales_int4, dst_scales, src_scales, ld_nc_global, st_na_global);
        }
    }
}

void dispatch_fp4(void* packed_recv_x, void* packed_recv_x_scales,
                int* packed_recv_src_info, int64_t* packed_recv_layout_range,
                int* packed_recv_count,
                int* cumulative_local_expert_recv_stats,
                void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
                const void* x, const void* x_scales, const int* topk_idx,
                int* next_clean, int num_next_clean_int,
                int num_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
                int num_topk, int num_experts, int rank, int num_ranks,
                void* workspace, int num_device_sms,
                hipStream_t stream, int phases) {
    constexpr int kNumMaxTopK = 9;
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = 32 / num_warp_groups;
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0);
    EP_HOST_ASSERT(kNumMaxTopK + 1 <= num_warp_groups * num_warps_per_group);

    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_sms = ceil_div(num_experts, num_warp_groups);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopK);

    // Workspace checks
    auto atomic_counter_per_expert = static_cast<int*>(workspace);
    auto atomic_finish_counter_per_expert = atomic_counter_per_expert + num_experts;
    EP_HOST_ASSERT(num_experts * sizeof(int) * 2 <= NUM_WORKSPACE_BYTES);
#define SWITCH_HIDDEN_FP4(case_macro) \
switch (hidden) { \
    case 4096: case_macro(4096); \
    case 7168: case_macro(7168); \
    default: EP_HOST_ASSERT(false && "Unsupported hidden"); \
} while (false)

#define DISPATCH_LAUNCH_CASE(hidden) { \
auto dispatch_func = dispatch_fp4<hidden>; \
LAUNCH_KERNEL(&cfg, dispatch_func, \
                packed_recv_x, packed_recv_x_scales, \
                packed_recv_src_info, packed_recv_layout_range, \
                packed_recv_count, \
                cumulative_local_expert_recv_stats, \
                rdma_recv_x, rdma_recv_count, rdma_x, \
                x, x_scales, topk_idx, \
                atomic_counter_per_expert, atomic_finish_counter_per_expert, \
                next_clean, num_next_clean_int, \
                num_tokens, num_max_dispatch_tokens_per_rank, \
                num_topk, num_experts, rank, num_ranks, \
                num_warp_groups, num_warps_per_group, \
                phases); } break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);
    SWITCH_HIDDEN_FP4(DISPATCH_LAUNCH_CASE);
#undef DISPATCH_LAUNCH_CASE
#undef SWITCH_HIDDEN_FP4
}
}
}